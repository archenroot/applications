#define ARRAY_CHECKS

#ifndef NAN

#include <math_constants.h>

#define NAN CUDART_NAN

#endif

#ifndef INFINITY

#include <math_constants.h>

#define INFINITY CUDART_INF

#endif


#include <hip/hip_runtime.h>
#include <stdio.h>

__shared__ size_t m_Local[3];

__shared__ char m_shared[40960];

__device__
int getThreadId(){
  return blockIdx.x * blockDim.x + threadIdx.x;
}
__device__
int getThreadIdxx(){
  return threadIdx.x;
}
__device__
int getBlockDimx(){
  return blockDim.x;
}
__device__
void edu_syr_pcpratts_syncthreads(){
  __syncthreads();
}
__device__
void edu_syr_pcpratts_threadfence(){
  __threadfence();
}
__device__ clock_t global_now;

/*HAMA_PIPES_HEADER_CODE_IGNORE_IN_TWEAKS_START*/

/* before HostDeviceInterface
nvcc generated.cu --ptxas-options=-v
ptxas info    : 8 bytes gmem, 4 bytes cmem[14]
ptxas info    : Compiling entry function '_Z5entryPcS_PiPxS1_S0_S0_i' for 'sm_10'
ptxas info    : Used 5 registers, 104 bytes smem, 20 bytes cmem[1]

after HostDeviceInterface

nvcc generated.cu --ptxas-options=-v

ptxas info    : 72 bytes gmem, 36 bytes cmem[14]
ptxas info    : Compiling entry function '_Z5entryPcS_PiPxS1_S0_S0_iS0_' for 'sm_10'
ptxas info    : Used 5 registers, 112 bytes smem, 20 bytes cmem[1]


nvcc generated.cu --ptxas-options=-v -arch sm_20

ptxas info    : 72 bytes gmem, 72 bytes cmem[14]
ptxas info    : Compiling entry function '_Z5entryPcS_PiPxS1_S0_S0_iS0_' for 'sm_20'
ptxas info    : Function properties for _Z5entryPcS_PiPxS1_S0_S0_iS0_
    0 bytes stack frame, 0 bytes spill stores, 0 bytes spill loads
ptxas info    : Used 12 registers, 24 bytes smem, 104 bytes cmem[0]

*/

#include <string>

#define STR_SIZE 1024

using std::string;

class HostDeviceInterface {
public:
  volatile bool is_debugging; 

  // Only one thread is able to use the
  // HostDeviceInterface
  volatile int lock_thread_id; 

  // HostMonitor has_task
  volatile bool has_task;

  // HostMonitor is done (end of communication)
  volatile bool done;

  // Request for HostMonitor
  enum MESSAGE_TYPE {
    START_MESSAGE, SET_BSPJOB_CONF, SET_INPUT_TYPES,
    RUN_SETUP, RUN_BSP, RUN_CLEANUP,
    READ_KEYVALUE, WRITE_KEYVALUE,
    GET_MSG, GET_MSG_COUNT,
    SEND_MSG, SYNC,
    GET_ALL_PEERNAME, GET_PEERNAME,
    GET_PEER_INDEX, GET_PEER_COUNT, GET_SUPERSTEP_COUNT,
    REOPEN_INPUT, CLEAR,
    CLOSE, ABORT,
    DONE, TASK_DONE,
    REGISTER_COUNTER, INCREMENT_COUNTER,
    SEQFILE_OPEN, SEQFILE_READNEXT,
    SEQFILE_APPEND, SEQFILE_CLOSE,
    PARTITION_REQUEST, PARTITION_RESPONSE,
    LOG, END_OF_DATA,
    UNDEFINED
  };
  volatile MESSAGE_TYPE command;

  // Command parameter
  volatile bool use_int_val1; // in int_val1
  volatile bool use_int_val2; // in int_val2
  volatile bool use_int_val3; // in int_val3
  volatile bool use_long_val1; // in long_val1
  volatile bool use_long_val2; // in long_val2
  volatile bool use_float_val1; // in float_val1
  volatile bool use_float_val2; // in float_val2
  volatile bool use_double_val1; // in double_val1
  volatile bool use_double_val2; // in double_val2
  volatile bool use_str_val1; // in str_val1
  volatile bool use_str_val2; // in str_val2
  volatile bool use_str_val3; // in str_val3

  // Transfer variables (used in sendCommand and getResult)
  volatile int int_val1;
  volatile int int_val2;
  volatile int int_val3;
  volatile long long_val1;
  volatile long long_val2;
  volatile float float_val1;
  volatile float float_val2;
  volatile double double_val1;
  volatile double double_val2;
  volatile char str_val1[STR_SIZE];
  volatile char str_val2[STR_SIZE];
  volatile char str_val3[255];

  enum TYPE {
    INT, LONG, FLOAT, DOUBLE, STRING, STRING_ARRAY,
    KEY_VALUE_PAIR, NULL_TYPE, NOT_AVAILABLE
  };
  volatile TYPE return_type;
  volatile TYPE key_type;
  volatile TYPE value_type;

  volatile bool end_of_data;

  // Response of HostMonitor
  volatile bool is_result_available;

  HostDeviceInterface() {
    init();
  }

  void init() {
    is_debugging = false;
    lock_thread_id = -1;
    has_task = false;
    done = false;
    command = UNDEFINED;
    use_int_val1 = false;
    use_int_val2 = false;
    use_int_val3 = false;
    use_long_val1 = false;
    use_long_val2 = false;
    use_float_val1 = false;
    use_float_val2 = false;
    use_double_val1 = false;
    use_double_val2 = false;
    use_str_val1 = false;
    use_str_val2 = false;
    use_str_val3 = false;
    int_val1 = 0;
    int_val2 = 0;
    int_val3 = 0;
    long_val1 = 0;
    long_val2 = 0;
    float_val1 = 0;
    float_val2 = 0;
    double_val1 = 0;
    double_val2 = 0;
    key_type = NOT_AVAILABLE;
    value_type = NOT_AVAILABLE;
    end_of_data = true;
    is_result_available = false;
  }

  ~HostDeviceInterface() {}
};

__device__ HostDeviceInterface *host_device_interface;

/*HAMA_PIPES_HEADER_CODE_IGNORE_IN_TWEAKS_END*/
__device__ double java_lang_Math_abs8_8_( char * gc_info, double parameter0, int * exception);

__device__ int java_lang_Float_toString9_7_( char * gc_info, float parameter0, int * exception);

__device__ int java_lang_Object_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception);

__device__ void java_lang_Object_initab850b60f96d11de8a390800200c9a66_body0_( char * gc_info, int thisref, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception);

__device__ void at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_StringBuilder_append10_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ double java_lang_Math_sqrt8_8_( char * gc_info, double parameter0, int * exception);

__device__ void at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_gpuMethod0_( char * gc_info, int thisref, int * exception);

__device__ double double__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void double__array_set( char * gc_info, int thisref, int parameter0, double parameter1, int * exception);

__device__ int double__array_new( char * gc_info, int size, int * exception);

__device__ int double__array_new_multi_array( char * gc_info, int dim0, int * exception);

__device__ int int__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void int__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int int__array_new( char * gc_info, int size, int * exception);

__device__ int int__array_new_multi_array( char * gc_info, int dim0, int * exception);

__device__ int edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_initab850b60f96d11de8a390800200c9a660_13_13_( char * gc_info, int parameter0, int parameter1, int * exception);

__device__ void edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_initab850b60f96d11de8a390800200c9a66_body0_13_13_( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int java_lang_AbstractStringBuilder_initab850b60f96d11de8a390800200c9a660_5_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_AbstractStringBuilder_initab850b60f96d11de8a390800200c9a66_body0_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void java_lang_Integer_getChars0_5_5_a14_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception);

__device__ void java_lang_Long_getChars0_6_5_a14_( char * gc_info, long long parameter0, int parameter1, int parameter2, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_new( char * gc_info, int size, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_new_multi_array( char * gc_info, int dim0, int * exception);

__device__ int java_lang_Double_toString9_8_( char * gc_info, double parameter0, int * exception);

__device__ int java_lang_Integer_toString9_5_( char * gc_info, int parameter0, int * exception);

__device__ int java_lang_String_length5_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_Exception_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_Exception_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int invoke_java_lang_StringBuilder_toString9_( char * gc_info, int thisref, int * exception);

__device__ int invoke_java_lang_Object_toString9_( char * gc_info, int thisref, int * exception);

__device__ int invoke_java_lang_Object_toString9_( char * gc_info, int thisref, int * exception);

__device__ int invoke_java_lang_StringBuilder_toString9_( char * gc_info, int thisref, int * exception);

__device__ void at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_add0_11_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_AbstractStringBuilder_append15_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int invoke_java_lang_AbstractStringBuilder_append15_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int invoke_java_lang_StringBuilder_append15_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int edu_syr_pcpratts_rootbeer_runtime_Sentinal_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception);

__device__ void edu_syr_pcpratts_rootbeer_runtime_Sentinal_initab850b60f96d11de8a390800200c9a66_body0_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_String__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void java_lang_String__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int java_lang_String__array_new( char * gc_info, int size, int * exception);

__device__ int java_lang_String__array_new_multi_array( char * gc_info, int dim0, int * exception);

__device__ long long instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_converged( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_converged( char * gc_info, int thisref, long long parameter0, int * exception);

__device__ int static_getter_java_lang_Integer_DigitOnes( char * gc_info, int * exception);

__device__ void static_setter_java_lang_Integer_DigitOnes( char * gc_info, int parameter0, int * expcetion);

__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int static_getter_java_lang_Integer_DigitTens( char * gc_info, int * exception);

__device__ void static_setter_java_lang_Integer_DigitTens( char * gc_info, int parameter0, int * expcetion);

__device__ int instance_getter_java_lang_String_hash( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_String_hash( char * gc_info, int thisref, int parameter0, int * exception);

__device__ long long instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_superstepCount( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_superstepCount( char * gc_info, int thisref, long long parameter0, int * exception);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_maxIterations( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_maxIterations( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_Class_name( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_Class_name( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_cache( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_cache( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_AbstractStringBuilder_value( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_AbstractStringBuilder_value( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int static_getter_java_lang_AbstractStringBuilder_sizeTable( char * gc_info, int * exception);

__device__ void static_setter_java_lang_AbstractStringBuilder_sizeTable( char * gc_info, int parameter0, int * expcetion);

__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayLength( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayLength( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int static_getter_java_lang_Integer_digits( char * gc_info, int * exception);

__device__ void static_setter_java_lang_Integer_digits( char * gc_info, int parameter0, int * expcetion);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_centers( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_centers( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_AbstractStringBuilder_count( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_AbstractStringBuilder_count( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int static_getter_java_lang_System_out( char * gc_info, int * exception);

__device__ void static_setter_java_lang_System_out( char * gc_info, int parameter0, int * expcetion);

__device__ int instance_getter_java_lang_Throwable_detailMessage( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_Throwable_detailMessage( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int static_getter_java_lang_Integer_sizeTable( char * gc_info, int * exception);

__device__ void static_setter_java_lang_Integer_sizeTable( char * gc_info, int parameter0, int * expcetion);

__device__ int instance_getter_java_lang_String_count( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_String_count( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_String_value( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_String_value( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayIndex( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayIndex( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_String_offset( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_String_offset( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_Throwable_cause( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_Throwable_cause( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_array( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_array( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception);

__device__ void java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a66_body0_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_IllegalArgumentException_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_IllegalArgumentException_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_String_initab850b60f96d11de8a390800200c9a66( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_String_initab850b60f96d11de8a390800200c9a66_body( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_RuntimeException_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_RuntimeException_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_Object_hashCode( char * gc_info, int thisref, int * exception);

__device__ int java_lang_StringBuilder_toString9_( char * gc_info, int thisref, int * exception);

__device__ void at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_assignCenters0_5_5_5_5_5_( char * gc_info, int thisref, int parameter0, int parameter1, int parameter2, int parameter3, int parameter4, int * exception);

__device__ int edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_getKey13_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_AbstractStringBuilder_append15_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_IndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_IndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int invoke_java_lang_String_hashCode5_( char * gc_info, int thisref, int * exception);

__device__ int invoke_java_lang_Object_hashCode( char * gc_info, int thisref, int * exception);

__device__ int java_lang_Long_toString9_6_( char * gc_info, long long parameter0, int * exception);

__device__ int java_lang_String_initab850b60f96d11de8a390800200c9a660_5_5_a14_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception);

__device__ void java_lang_String_initab850b60f96d11de8a390800200c9a66_body0_5_5_a14_( char * gc_info, int thisref, int parameter0, int parameter1, int parameter2, int * exception);

__device__ int java_lang_String_hashCode5_( char * gc_info, int thisref, int * exception);

__device__ void java_lang_String_getChars0_5_5_a14_5_( char * gc_info, int thisref, int parameter0, int parameter1, int parameter2, int parameter3, int * exception);

__device__ int java_lang_Throwable_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_Throwable_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_Object_toString9_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_String_initab850b60f96d11de8a390800200c9a660_a14_5_5_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception);

__device__ void java_lang_String_initab850b60f96d11de8a390800200c9a66_body0_a14_5_5_( char * gc_info, int thisref, int parameter0, int parameter1, int parameter2, int * exception);

__device__ void java_lang_AbstractStringBuilder_expandCapacity0_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int invoke_java_lang_AbstractStringBuilder_append15_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int invoke_java_lang_StringBuilder_append15_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_Integer_stringSize5_5_( char * gc_info, int parameter0, int * exception);

__device__ int edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception);

__device__ int invoke_java_lang_String_hashCode5_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_Math_min5_5_5_( char * gc_info, int parameter0, int parameter1, int * exception);

__device__ char char__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void char__array_set( char * gc_info, int thisref, int parameter0, char parameter1, int * exception);

__device__ int char__array_new( char * gc_info, int size, int * exception);

__device__ int char__array_new_multi_array( char * gc_info, int dim0, int * exception);

__device__ int java_lang_Integer_toHexString9_5_( char * gc_info, int parameter0, int * exception);

__device__ int java_lang_Integer_toUnsignedString9_5_5_( char * gc_info, int parameter0, int parameter1, int * exception);

__device__ int java_lang_Long_stringSize5_6_( char * gc_info, long long parameter0, int * exception);

__device__ int java_util_Arrays_copyOfRangea14_a14_5_5_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception);

__device__ int java_lang_Boolean_toString9_1_( char * gc_info, char parameter0, int * exception);

__device__ void 
java_lang_System_arraycopy( char * gc_info, int src_handle, int srcPos, int dest_handle, int destPos, int length, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_getLength5_( char * gc_info, int thisref, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_getNearestCenter5_5_5_5_( char * gc_info, int thisref, int parameter0, int parameter1, int parameter2, int * exception);

__device__ int java_lang_StringBuilder_append10_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception);

__device__ void edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_initab850b60f96d11de8a390800200c9a66_body0_( char * gc_info, int thisref, int * exception);

__device__ double at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_measureEuclidianDistance8_5_5_5_( char * gc_info, int thisref, int parameter0, int parameter1, int parameter2, int * exception);

__device__ int java_util_Arrays_copyOfa14_a14_5_( char * gc_info, int parameter0, int parameter1, int * exception);

__device__ int double__array__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void double__array__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int double__array__array_new( char * gc_info, int size, int * exception);

__device__ int double__array__array_new_multi_array( char * gc_info, int dim0, int dim1, int * exception);

__device__ int java_lang_Character_toString9_3_( char * gc_info, char parameter0, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception);

__device__ void at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_initab850b60f96d11de8a390800200c9a66_body0_( char * gc_info, int thisref, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_get11_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_AbstractStringBuilder_stringSizeOfInt5_5_( char * gc_info, int parameter0, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_toArraya12_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_StringIndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a660_5_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_StringIndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a66_body0_5_( char * gc_info, int thisref, int parameter0, int * exception);

#define GC_OBJ_TYPE_COUNT char

#define GC_OBJ_TYPE_COLOR char

#define GC_OBJ_TYPE_TYPE int

#define GC_OBJ_TYPE_CTOR_USED char

#define GC_OBJ_TYPE_SIZE int

#define COLOR_GREY 0

#define COLOR_BLACK 1

#define COLOR_WHITE 2

__device__ void edu_syr_pcpratts_gc_collect( char * gc_info);

__device__ void edu_syr_pcpratts_gc_assign( char * gc_info, int * lhs, int rhs);

__device__  char * edu_syr_pcpratts_gc_deref( char * gc_info, int handle);

__device__ int edu_syr_pcpratts_gc_malloc( char * gc_info, int size);

__device__ unsigned long long edu_syr_pcpratts_gc_malloc_no_fail( char * gc_info, int size);

__device__ int edu_syr_pcpratts_classConstant(int type_num);

__device__ long long java_lang_System_nanoTime( char * gc_info, int * exception);

#define CACHE_SIZE_BYTES 32

#define CACHE_SIZE_INTS (CACHE_SIZE_BYTES / sizeof(int))

#define CACHE_ENTRY_SIZE 4

#define TO_SPACE_OFFSET               0

#define TO_SPACE_FREE_POINTER_OFFSET  8

__device__
void edu_syr_pcpratts_exitMonitorMem( char * gc_info, char * mem, int old){
  if(old == -1){
   
    edu_syr_pcpratts_threadfence(); 
    atomicExch((int *) mem, -1);
  }
}
__device__ double java_lang_StrictMath_sqrt( char * gc_info , double parameter0 , int * exception ) {
 
  return sqrt(parameter0); 
}
__device__ 
int edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getThreadId( char * gc_info, int * exception){
  return getThreadId();
}
__device__ 
int edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getThreadIdxx( char * gc_info, int * exception){
  return getThreadIdxx();
}
__device__ 
int edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getBlockDimx( char * gc_info, int * exception){
  return getBlockDimx();
}
__device__
int edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedInteger( char * gc_info, int index, int * exception){
#ifdef ARRAY_CHECKS
  if(index < 0 || index + 4 >= 40960){
    *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, 
      index, 0, 40960, exception);
    return 0;
  }
#endif
  int ret = m_shared[index] & 0x000000ff;
  ret |= (m_shared[index + 1] << 8)  & 0x0000ff00;
  ret |= (m_shared[index + 2] << 16) & 0x00ff0000;
  ret |= (m_shared[index + 3] << 24) & 0xff000000; 
  return ret;
}
__device__
void edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedInteger( char * gc_info, int index, int value, int * exception){
  
#ifdef ARRAY_CHECKS
  if(index < 0 || index + 4 >= 40960){
    *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, 
      index, 0, 40960, exception);
    return;
  }
#endif
  m_shared[index] = (char) (value & 0xff);
  m_shared[index + 1] = (char) ((value >> 8)  & 0xff);
  m_shared[index + 2] = (char) ((value >> 16) & 0xff);
  m_shared[index + 3] = (char) ((value >> 24) & 0xff);
}
__device__
long long edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedLong( char * gc_info, int index, int * exception){
#ifdef ARRAY_CHECKS
  if(index < 0 || index + 8 >= 40960){
    *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, 
      index, 0, 40960, exception);
    return 0;
  }
#endif
  long long ret = 0;
  ret |=  ((long long) m_shared[index]) & 0x00000000000000ffL;
  ret |= ((long long) m_shared[index + 1] << 8)  & 0x000000000000ff00L;
  ret |= ((long long) m_shared[index + 2] << 16) & 0x0000000000ff0000L;
  ret |= ((long long) m_shared[index + 3] << 24) & 0x00000000ff000000L;
  ret |= ((long long) m_shared[index + 4] << 32) & 0x000000ff00000000L;
  ret |= ((long long) m_shared[index + 5] << 40) & 0x0000ff0000000000L;
  ret |= ((long long) m_shared[index + 6] << 48) & 0x00ff000000000000L;
  ret |= ((long long) m_shared[index + 7] << 56) & 0xff00000000000000L;
  return ret;
}
__device__
void edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedLong( char * gc_info, int index, long long value, int * exception){
#ifdef ARRAY_CHECKS
  if(index < 0 || index + 8 >= 40960){
    *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, 
      index, 0, 40960, exception);
    return;
  }
#endif
  m_shared[index] = (char) (value & 0x00000000000000ffL);
  m_shared[index + 1] = (char) ((value >> 8)  & 0x00000000000000ffL);
  m_shared[index + 2] = (char) ((value >> 16) & 0x00000000000000ffL);
  m_shared[index + 3] = (char) ((value >> 24) & 0x00000000000000ffL);
  m_shared[index + 4] = (char) ((value >> 32) & 0x00000000000000ffL);
  m_shared[index + 5] = (char) ((value >> 40) & 0x00000000000000ffL);
  m_shared[index + 6] = (char) ((value >> 48) & 0x00000000000000ffL);
  m_shared[index + 7] = (char) ((value >> 56) & 0x00000000000000ffL);
}
__device__
double edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedDouble( char * gc_info, int index, int * exception){
  long long long_value = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedLong(gc_info, index, exception);
  return *((double *) &long_value);
}
__device__
void edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedDouble( char * gc_info, int index, double value, int * exception){
  long long long_value = *((long long *) &value);
  edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedLong(gc_info, index, long_value, exception);
}
__device__
void java_io_PrintStream_println0_9_( char * gc_info, int thisref, int str_ret, int * exception){
  int valueref;
  int count;
  int offset;
  int i;
  int curr_offset;
  char * valueref_deref;
  valueref = instance_getter_java_lang_String_value(gc_info, str_ret, exception);  
  if(*exception != 0){
    return; 
  }
 
  count = instance_getter_java_lang_String_count(gc_info, str_ret, exception);
  if(*exception != 0){
    return; 
  }
 
  offset = instance_getter_java_lang_String_offset(gc_info, str_ret, exception);
  if(*exception != 0){
    return; 
  }
 
  valueref_deref = (char *) edu_syr_pcpratts_gc_deref(gc_info, valueref);
  for(i = offset; i < count; ++i){
    curr_offset = 32 + (i * 4);
    printf("%c", valueref_deref[curr_offset]);
  }
  printf("\n");
}
__device__
void java_io_PrintStream_println0_6_( char * gc_info, int thisref, long long value, int * exception){
  printf("%lld\n", value);
}
__device__
void java_io_PrintStream_println0_8_( char * gc_info, int thisref, double value, int * exception){
  printf("%e\n", value);
}
__device__
void java_io_PrintStream_print0_9_( char * gc_info, int thisref, int str_ret, int * exception){
  int valueref;
  int count;
  int offset;
  int i;
  int curr_offset;
  char * valueref_deref;
  valueref = instance_getter_java_lang_String_value(gc_info, str_ret, exception);  
  if(*exception != 0){
    return; 
  }
 
  count = instance_getter_java_lang_String_count(gc_info, str_ret, exception);
  if(*exception != 0){
    return; 
  }
 
  offset = instance_getter_java_lang_String_offset(gc_info, str_ret, exception);
  if(*exception != 0){
    return; 
  }
 
  valueref_deref = (char *) edu_syr_pcpratts_gc_deref(gc_info, valueref);
  for(i = offset; i < count; ++i){
    curr_offset = 32 + (i * 4);
    printf("%c", valueref_deref[curr_offset]);
  }
}
__device__
void java_io_PrintStream_print0_5_( char * gc_info, int thisref, int value, int * exception){
  printf("%d", value);
}
__device__ 
void edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_syncthreads( char * gc_info, int * exception){
  edu_syr_pcpratts_syncthreads();
}
__device__ char
edu_syr_pcpratts_cmp(long long lhs, long long rhs){
  if(lhs > rhs)
    return 1;
  if(lhs < rhs)
    return -1;
  return 0;
}
__device__ char
edu_syr_pcpratts_cmpl(double lhs, double rhs){
  if(lhs > rhs)
    return 1;
  if(lhs < rhs)
    return -1;
  if(lhs == rhs)
    return 0;
  return -1;
}
__device__ char
edu_syr_pcpratts_cmpg(double lhs, double rhs){
  if(lhs > rhs)
    return 1;
  if(lhs < rhs)
    return -1;
  if(lhs == rhs)
    return 0;
  return 1;
}
__device__ void
edu_syr_pcpratts_gc_set_count( char * mem_loc, GC_OBJ_TYPE_COUNT value){
  mem_loc[0] = value;
}
__device__ void
edu_syr_pcpratts_gc_set_color( char * mem_loc, GC_OBJ_TYPE_COLOR value){
  mem_loc += sizeof(GC_OBJ_TYPE_COUNT);
  mem_loc[0] = value;
}
__device__ void
edu_syr_pcpratts_gc_init_monitor( char * mem_loc){
  int * addr;
  mem_loc += 16;
  addr = (int *) mem_loc;
  *addr = -1;
}
__device__ void
edu_syr_pcpratts_gc_set_type( char * mem_loc, GC_OBJ_TYPE_TYPE value){
  mem_loc += sizeof(GC_OBJ_TYPE_COUNT) + sizeof(GC_OBJ_TYPE_COLOR) + sizeof(char) +
    sizeof(GC_OBJ_TYPE_CTOR_USED);
  *(( GC_OBJ_TYPE_TYPE *) &mem_loc[0]) = value;
}
__device__ GC_OBJ_TYPE_TYPE
edu_syr_pcpratts_gc_get_type( char * mem_loc){
  mem_loc += sizeof(GC_OBJ_TYPE_COUNT) + sizeof(GC_OBJ_TYPE_COLOR) + sizeof(char) +
    sizeof(GC_OBJ_TYPE_CTOR_USED);
  return *(( GC_OBJ_TYPE_TYPE *) &mem_loc[0]);
}
__device__ void
edu_syr_pcpratts_gc_set_ctor_used( char * mem_loc, GC_OBJ_TYPE_CTOR_USED value){
  mem_loc += sizeof(GC_OBJ_TYPE_COUNT) + sizeof(GC_OBJ_TYPE_COLOR) + sizeof(char);
  mem_loc[0] = value;
}
__device__ void
edu_syr_pcpratts_gc_set_size( char * mem_loc, GC_OBJ_TYPE_SIZE value){
  mem_loc += sizeof(GC_OBJ_TYPE_COUNT) + sizeof(GC_OBJ_TYPE_COLOR) + sizeof(char) + 
    sizeof(GC_OBJ_TYPE_CTOR_USED) + sizeof(GC_OBJ_TYPE_TYPE);
  *(( GC_OBJ_TYPE_SIZE *) &mem_loc[0]) = value;
}
__device__ int edu_syr_pcpratts_getint( char * buffer, int pos){
  return *(( int *) &buffer[pos]);
}
__device__ void edu_syr_pcpratts_setint( char * buffer, int pos, int value){
  *(( int *) &buffer[pos]) = value;
}
__device__ int
edu_syr_pcpratts_strlen(char * str_constant){
  int ret = 0;
  while(1){
    if(str_constant[ret] != 
'\0'
){
      ret++;
    }
 else {
      return ret;
    }
  }
}
__device__ int
edu_syr_pcpratts_array_length( char * gc_info, int thisref){
  
  
  
  
  
     char * thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
    int ret = edu_syr_pcpratts_getint(thisref_deref, 12);
    return ret;
  
}
__device__
int java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(char * gc_info, int * exception){
 
  int thisref;
  char * thisref_deref;
  int chars;
  thisref = edu_syr_pcpratts_gc_malloc(gc_info , 48);
  if(thisref == -1){
    *exception = 21352; 
    return -1; 
  }
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  edu_syr_pcpratts_gc_set_count(thisref_deref, 1); 
  edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY); 
  edu_syr_pcpratts_gc_set_type(thisref_deref, 2906); 
  edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1); 
  edu_syr_pcpratts_gc_set_size(thisref_deref, 48); 
  edu_syr_pcpratts_gc_init_monitor(thisref_deref); 
  chars = char__array_new(gc_info, 0, exception);
  instance_setter_java_lang_AbstractStringBuilder_value(gc_info, thisref, chars, exception); 
  instance_setter_java_lang_AbstractStringBuilder_count(gc_info, thisref, 0, exception);
  return thisref; 
}
__device__
int java_lang_String_initab850b60f96d11de8a390800200c9a66(char * gc_info, int parameter0, int * exception) {
 
  int r0 = -1; 
  int r1 = -1; 
  int i0; 
  int $r2 = -1; 
  int thisref; 
  char * thisref_deref; 
  int i;
  int len;
  int characters_copy;
  char ch;
  
  thisref = -1; 
  edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 48)); 
  if(thisref == -1) {
 
    *exception = 21352; 
    return -1; 
  }
 
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref); 
  edu_syr_pcpratts_gc_set_count(thisref_deref, 1); 
  edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY); 
  edu_syr_pcpratts_gc_set_type(thisref_deref, 2906); 
  edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1); 
  edu_syr_pcpratts_gc_set_size(thisref_deref, 48); 
  edu_syr_pcpratts_gc_init_monitor(thisref_deref); 
  len = edu_syr_pcpratts_array_length(gc_info, parameter0);
  characters_copy = char__array_new(gc_info, len, exception);
  for(i = 0; i < len; ++i){
    ch = char__array_get(gc_info, parameter0, i, exception);
    char__array_set(gc_info, characters_copy, i, ch, exception);
  }
  instance_setter_java_lang_String_value(gc_info, thisref, characters_copy, exception); 
  instance_setter_java_lang_String_count(gc_info, thisref, len, exception); 
  instance_setter_java_lang_String_offset(gc_info, thisref, 0, exception); 
  return thisref; 
}
__device__ int 
char__array_new( char * gc_info, int size, int * exception);

__device__ void 
char__array_set( char * gc_info, int thisref, int parameter0, char parameter1, int * exception);

__device__ int
edu_syr_pcpratts_string_constant( char * gc_info, char * str_constant, int * exception){
  int i;
  int len = edu_syr_pcpratts_strlen(str_constant);
  int characters = char__array_new(gc_info, len, exception);
  unsigned long long * addr = (unsigned long long *) (gc_info + TO_SPACE_FREE_POINTER_OFFSET);
  for(i = 0; i < len; ++i){
    char__array_set(gc_info, characters, i, str_constant[i], exception);
  }
  return java_lang_String_initab850b60f96d11de8a390800200c9a66(gc_info, characters, exception);
}
__device__ void
edu_syr_pcpratts_gc_assign( char * gc_info, int * lhs_ptr, int rhs){
  *lhs_ptr = rhs;
}
__device__ int java_lang_StackTraceElement__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void java_lang_StackTraceElement__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int java_lang_StackTraceElement__array_new( char * gc_info, int size, int * exception);

__device__ int java_lang_StackTraceElement_initab850b60f96d11de8a390800200c9a660_3_3_3_4_( char * gc_info, int parameter0, int parameter1, int parameter2, int parameter3, int * exception);

__device__ void instance_setter_java_lang_RuntimeException_stackDepth( char * gc_info, int thisref, int parameter0);

__device__ int instance_getter_java_lang_RuntimeException_stackDepth( char * gc_info, int thisref);

__device__ int java_lang_StackTraceElement__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_Throwable_stackTrace( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_Throwable_stackTrace( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_Throwable_fillInStackTrace( char * gc_info, int thisref, int * exception){
  
  
  return thisref;
}
__device__ void instance_setter_java_lang_Throwable_cause( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void instance_setter_java_lang_Throwable_detailMessage( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void instance_setter_java_lang_Throwable_stackDepth( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void java_lang_VirtualMachineError_initab850b60f96d11de8a390800200c9a66_body0_( char * gc_info, int thisref, int * exception);

__device__ int
java_lang_Object_hashCode( char * gc_info, int thisref, int * exception){
  return thisref;
}
__device__ int
java_lang_Class_getName( char * gc_info , int thisref , int * exception ) {
 
  int $r1 =-1 ; 
  $r1 = instance_getter_java_lang_Class_name ( gc_info , thisref , exception ) ; 
  if ( * exception != 0 ) {
 
    return 0 ; 
  }
 
  return $r1;
}
__device__ int
java_lang_Object_getClass( char * gc_info , int thisref, int * exception ) {
 
  char * mem_loc = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  int type = edu_syr_pcpratts_gc_get_type(mem_loc);
  return edu_syr_pcpratts_classConstant(type);
}
__device__ 
int java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a6610_9_(char * gc_info, 
  int str ,int * exception){
 
  int r0 = -1; 
  int thisref; 
  int str_value;
  int str_count;  
  char * thisref_deref; 
  thisref = -1;
  edu_syr_pcpratts_gc_assign ( gc_info , & thisref , edu_syr_pcpratts_gc_malloc ( gc_info , 48 ) ) ; 
  if ( thisref ==-1 ) {
 
    * exception = 21352; 
    return-1 ; 
  }
 
  thisref_deref = edu_syr_pcpratts_gc_deref ( gc_info , thisref ) ; 
  edu_syr_pcpratts_gc_set_count ( thisref_deref , 0 ) ; 
  edu_syr_pcpratts_gc_set_color ( thisref_deref , COLOR_GREY ) ; 
  edu_syr_pcpratts_gc_set_type ( thisref_deref , 16901 ) ; 
  edu_syr_pcpratts_gc_set_ctor_used ( thisref_deref , 1 ) ; 
  edu_syr_pcpratts_gc_set_size ( thisref_deref , 44 ) ; 
  edu_syr_pcpratts_gc_init_monitor ( thisref_deref ) ; 
  str_value = instance_getter_java_lang_String_value(gc_info, str, exception);
  str_count = instance_getter_java_lang_String_count(gc_info, str, exception);
  instance_setter_java_lang_AbstractStringBuilder_value(gc_info, thisref, str_value, exception); 
  instance_setter_java_lang_AbstractStringBuilder_count(gc_info, thisref, str_count, exception); 
  return thisref; 
}
__device__ 
int java_lang_StringBuilder_append10_9_(char * gc_info, int thisref,
  int parameter0, int * exception){
  int sb_value;
  int sb_count;
  int str_value;
  int str_count;
  int new_count;
  int new_sb_value;
  int i;
  char ch;
  int new_str;
  
  sb_value = instance_getter_java_lang_AbstractStringBuilder_value(gc_info, thisref,
    exception);
  sb_count = instance_getter_java_lang_AbstractStringBuilder_count(gc_info, thisref,
    exception);
  
  str_value = instance_getter_java_lang_String_value(gc_info, parameter0,
    exception);
  str_count = instance_getter_java_lang_String_count(gc_info, parameter0,
    exception);
  new_count = sb_count + str_count;
  new_sb_value = char__array_new(gc_info, new_count, exception);
  for(i = 0; i < sb_count; ++i){
    ch = char__array_get(gc_info, sb_value, i, exception);
    char__array_set(gc_info, new_sb_value, i, ch, exception);
  }
  for(i = 0; i < str_count; ++i){
    ch = char__array_get(gc_info, str_value, i, exception);
    char__array_set(gc_info, new_sb_value, sb_count + i, ch, exception);
  }
  
  new_str = java_lang_String_initab850b60f96d11de8a390800200c9a66(gc_info, 
    new_sb_value, exception);
  
  return java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a6610_9_(gc_info,
    new_str, exception);
}
__device__ 
int java_lang_StringBuilder_append10_5_(char * gc_info, int thisref,
  int parameter0, int * exception){
  int str = java_lang_Integer_toString9_5_(gc_info, parameter0, exception);
  return java_lang_StringBuilder_append10_9_(gc_info, thisref, str, exception);
}
__device__ 
int java_lang_StringBuilder_toString9_(char * gc_info, int thisref,
  int * exception){
 
  int value = instance_getter_java_lang_AbstractStringBuilder_value(gc_info, thisref,
    exception);
  return java_lang_String_initab850b60f96d11de8a390800200c9a66(gc_info, value, 
    exception);
}
/*****************************************************************************/
/* local methods */
__device__
int at_illecker_strlen(volatile char * str_constant) {
  int ret = 0;
  while(1) {
    if(str_constant[ret] != 
'\0'
) {
      ret++;
    }
 else {
      return ret;
    }
  }
}
__device__
int at_illecker_string_constant(char * gc_info, volatile char * str_constant, int * exception) {
  if (str_constant == 0) {
    return 0;
  }
  int i;
  int len = at_illecker_strlen(str_constant);
  int characters = char__array_new(gc_info, len, exception);
  
  if (host_device_interface->is_debugging) {
    printf("at_illecker_string_constant str: '"); 
  }
  for(i = 0; i < len; ++i) {
    char__array_set(gc_info, characters, i, str_constant[i], exception);
    if (host_device_interface->is_debugging) {
      printf("%c",str_constant[i]);
    }
  }
  if (host_device_interface->is_debugging) {
    printf("'\n");  
  }
  
  return java_lang_String_initab850b60f96d11de8a390800200c9a66(gc_info, characters, exception);
}
/*****************************************************************************/
/* toString methods */
__device__
double at_illecker_abs_val(double value) {
  double result = value;
  if (value < 0) {
    result = -value;
  }
  return result;
}
__device__
double at_illecker_pow10(int exp) {
  double result = 1;
  while (exp) {
    result *= 10;
    exp--;
  }
  return result;
}
__device__
long at_illecker_round(double value) {
  long intpart;
  intpart = value;
  value = value - intpart;
  if (value >= 0.5) {
    intpart++;
  }
  return intpart;
}
__device__
void at_illecker_set_char(char *buffer, int *currlen, int maxlen, char c) {
  if (*currlen < maxlen) {
    buffer[(*currlen)++] = c;
  }
}
__device__
int at_illecker_double_to_string(char * gc_info, double fvalue, int max, int * exception) {
  int signvalue = 0;
  double ufvalue;
  long intpart;
  long fracpart;
  char iconvert[20];
  char fconvert[20];
  int iplace = 0;
  int fplace = 0;
  int zpadlen = 0; 
  char buffer[64];
  int maxlen = 64;
  int currlen = 0;
  
  
  
  if (max < 0) {
    max = 6;
  }
  
  
  if (max > 9) {
    max = 9;
  }
  
  if (fvalue < 0) {
    signvalue = 
'-'
;
  }
  ufvalue = at_illecker_abs_val(fvalue);
  intpart = ufvalue;
  
  
  fracpart = at_illecker_round(at_illecker_pow10(max) * (ufvalue - intpart));
  if (fracpart >= at_illecker_pow10(max)) {
    intpart++;
    fracpart -= at_illecker_pow10(max);
  }
  
  
  
  do {
    iconvert[iplace++] = "0123456789abcdef"[intpart % 10];
    intpart = (intpart / 10);
  }
 while(intpart && (iplace < 20));
  if (iplace == 20) {
    iplace--;
  }
  iconvert[iplace] = 0;
  
  do {
    fconvert[fplace++] = "0123456789abcdef"[fracpart % 10];
    fracpart = (fracpart / 10);
  }
 while(fracpart && (fplace < 20));
  
  if (fplace == 20) {
    fplace--;
  }
  fconvert[fplace] = 0;
  
  zpadlen = max - fplace;
  if (zpadlen < 0) {
    zpadlen = 0;
  }
  
  
  
  if (signvalue) {
    at_illecker_set_char(buffer, &currlen, maxlen, signvalue);
  }
  
  while (iplace > 0) {
    at_illecker_set_char(buffer, &currlen, maxlen, iconvert[--iplace]);
  }
  
  if (max > 0) {
    
    
    
    at_illecker_set_char(buffer, &currlen, maxlen, 
'.'
);
    while (fplace > 0) {
      at_illecker_set_char(buffer, &currlen, maxlen, fconvert[--fplace]);
    }
  }
  
  while (zpadlen > 0) {
    at_illecker_set_char(buffer, &currlen, maxlen, 
'0'
);
    --zpadlen;
  }
  
  if (currlen < maxlen - 1) {
    buffer[currlen] = 
'\0'
;
  }
 else {
    buffer[maxlen - 1] = 
'\0'
;
  }
  return at_illecker_string_constant(gc_info, buffer, exception);
}
__device__ 
int java_lang_Double_toString9_8_(char * gc_info, double double_val, int * exception) {
  
  return at_illecker_double_to_string(gc_info, double_val, 6, exception);
}
/*****************************************************************************/
/* String.indexOf methods */
__device__
int at_illecker_strpos(char * gc_info, int str_value, int str_count, 
                       int sub_str_value, int sub_str_count, 
                       int start_pos, int * exception) {
  if ( (str_count == 0) || (sub_str_count == 0) || 
       (start_pos > str_count)) {
    return -1;
  }
  for (int i = start_pos; i < str_count; i++) {
    if (char__array_get(gc_info, str_value, i, exception) != 
        char__array_get(gc_info, sub_str_value, 0, exception)) {
      continue;
    }
    int found_pos = i;
    int found_sub_string = true;
    for (int j = 1; j < sub_str_count; j++) {
      i++;
      if (char__array_get(gc_info, str_value, i, exception) != 
          char__array_get(gc_info, sub_str_value, j, exception)) {
        found_sub_string = false;
        break;
      }
    }
    if (found_sub_string) {
      return found_pos;
    }
  }
  return -1;
}
/*****************************************************************************/
/* String.substring methods */
__device__
int at_illecker_substring(char * gc_info, int str_value, int str_count, 
                       int begin_index, int end_index, int * exception) {
  int new_length = 0;
  int new_string = -1;
  
  if (end_index == -1) {
 
    new_length = str_count - begin_index;
  }
 else {
    if (end_index < str_count) {
      new_length = end_index - begin_index;
    }
 else {
      new_length = str_count - begin_index;
    }
  }
 
  
  new_string = char__array_new(gc_info, new_length, exception);
  for(int i = 0; i < new_length; i++) {
    char__array_set(gc_info, new_string, i, char__array_get(gc_info, str_value, begin_index, exception), exception);
    begin_index++;
  }
  return java_lang_String_initab850b60f96d11de8a390800200c9a66(gc_info, new_string, exception);
}
/*****************************************************************************/
/* String.split methods */
__device__
int at_illecker_strcnt(char * gc_info, int str_value, int str_count, 
                       int sub_str_value, int sub_str_count, int * exception) {
  int occurrences = 0;
  if ( (str_count == 0) || (sub_str_count == 0) ) {
    return 0;
  }
  for (int i = 0; i < str_count; i++) {
    if (char__array_get(gc_info, str_value, i, exception) != 
        char__array_get(gc_info, sub_str_value, 0, exception)) {
      continue;
    }
    bool found_sub_string = true;
    for (int j = 1; j < sub_str_count; j++) {
      i++;
      if (char__array_get(gc_info, str_value, i, exception) != 
          char__array_get(gc_info, sub_str_value, j, exception)) {
        found_sub_string = false;
        break;
      }
    }
    if (found_sub_string) {
      occurrences++;
    }
  }
  return occurrences;
}
__device__
int at_illecker_split(char * gc_info, int str_obj_ref, int delim_str_obj_ref,
                      int limit, int * exception) {
  int return_obj = -1;
  int start = 0;
  int end = 0;
  int str_value = 0;
  int str_count = 0;
  int delim_str_value = 0;
  int delim_str_count = 0;
  int delim_occurrences = 0;
  str_value = instance_getter_java_lang_String_value(gc_info, str_obj_ref, exception);
  str_count = instance_getter_java_lang_String_count(gc_info, str_obj_ref, exception);
  delim_str_value = instance_getter_java_lang_String_value(gc_info, delim_str_obj_ref, exception);
  delim_str_count = instance_getter_java_lang_String_count(gc_info, delim_str_obj_ref, exception);
  
  delim_occurrences = at_illecker_strcnt(gc_info, str_value, str_count, 
                                         delim_str_value, delim_str_count, exception);
  
  if ( (limit <= 0) || (limit > delim_occurrences) ) {
    return_obj = java_lang_String__array_new(gc_info, delim_occurrences + 1, exception);
    limit = delim_occurrences + 1;
  }
 else {
    return_obj = java_lang_String__array_new(gc_info, limit, exception);
  }
  if (delim_occurrences == 0) {
    
    java_lang_String__array_set(gc_info, return_obj, 0, str_obj_ref, exception);
    
  }
 else {
    
    for (int i = 0; i < limit - 1; i++) {
      end = at_illecker_strpos(gc_info, str_value, str_count, 
                               delim_str_value, delim_str_count, start, exception);
      if (end == -1) {
        break;
      }
      
      java_lang_String__array_set(gc_info, return_obj, i,
        at_illecker_substring(gc_info, str_value, str_count, start, end, exception), exception);
      
      start = end + delim_str_count;
    }
    
    if (end != -1) {
      
      java_lang_String__array_set(gc_info, return_obj, limit - 1,
        at_illecker_substring(gc_info, str_value, str_count, start, -1, exception), exception);
    }
  }
  return return_obj;
}
__device__
int java_lang_String_split(char * gc_info, int str_obj_ref, int delim_str_obj_ref, int limit, int * exception) {
  return at_illecker_split(gc_info, str_obj_ref, delim_str_obj_ref, limit, exception);
}
__device__
int java_lang_String_split(char * gc_info, int str_obj_ref, int delim_str_obj_ref, int * exception) {
  return at_illecker_split(gc_info, str_obj_ref, delim_str_obj_ref, 0, exception);
}
/*****************************************************************************/
/* Parse methods */
__device__
bool at_illecker_is_digit(unsigned char c) {
  return ((c)>=
'0'
 && (c)<=
'9'
);
}
__device__
bool at_illecker_is_space(unsigned char c) {
  return ((c)==
' '
 || (c)==
'\f'
 || (c)==
'\n'
 || (c)==
'\r'
 || (c)==
'\t'
 || (c)==
'\v'
);
}
/* Argument1: String of ASCII digits, possibly
 * preceded by white space.  For bases
 * greater than 10, either lower- or
 * upper-case digits may be used.
 */
/* Argument2: Where to store address of terminating
 * character, or NULL.
 */
/* Argument3: Base for conversion.  Must be less
 * than 37.  If 0, then the base is chosen
 * from the leading characters of string:
 * "0x" means hex, "0" means octal, anything
 * else means decimal.
 */
__device__
unsigned long int at_illecker_strtoul(const char *string, char **end_ptr, int base) {
  register const char *p;
  register unsigned long int result = 0;
  register unsigned digit;
  int anyDigits = 0;
  int negative=0;
  int overflow=0;
  char cvtIn[] = {
    0, 1, 2, 3, 4, 5, 6, 7, 8, 9,		/* 
'0'
 - 
'9'
 */
    100, 100, 100, 100, 100, 100, 100,		/* punctuation */
    10, 11, 12, 13, 14, 15, 16, 17, 18, 19,	/* 
'A'
 - 
'Z'
 */
    20, 21, 22, 23, 24, 25, 26, 27, 28, 29,
    30, 31, 32, 33, 34, 35,
    100, 100, 100, 100, 100, 100,		/* punctuation */
    10, 11, 12, 13, 14, 15, 16, 17, 18, 19,	/* 
'a'
 - 
'z'
 */
    20, 21, 22, 23, 24, 25, 26, 27, 28, 29,
    30, 31, 32, 33, 34, 35
  }
;
  
  p = string;
  while (at_illecker_is_space((unsigned char) (*p))) {
    p += 1;
  }
  
  if (*p == 
'-'
) {
    negative = 1;
    p += 1;
  }
 else {
    if (*p == 
'+'
) {
      p += 1;
    }
  }
  
  
  if (base == 0) {
    if (*p == 
'0'
) {
      p += 1;
      if ((*p == 
'x'
) || (*p == 
'X'
)) {
        p += 1;
        base = 16;
      }
 else {
        
        
        anyDigits = 1;
        base = 8;
      }
    }
 else {
      base = 10;
    }
  }
 else if (base == 16) {
    
    if ((p[0] == 
'0'
) && ((p[1] == 
'x'
) || (p[1] == 
'X'
))) {
      p += 2;
    }
  }
  
  
  if (base == 8) {
    unsigned long maxres = 0xFFFFFFFFUL >> 3; 
    for ( ; ; p += 1) {
      digit = *p - 
'0'
;
      if (digit > 7) {
        break;
      }
      if (result > maxres) {
 
        overflow = 1;
      }
      result = (result << 3);
      if (digit > (0xFFFFFFFFUL - result)) {
 
        overflow = 1;
      }
      result += digit;
      anyDigits = 1;
    }
  }
 else if (base == 10) {
    unsigned long maxres = 0xFFFFFFFFUL / 10; 
    for ( ; ; p += 1) {
      digit = *p - 
'0'
;
      if (digit > 9) {
        break;
      }
      if (result > maxres) {
 
        overflow = 1;
      }
      result *= 10;
      if (digit > (0xFFFFFFFFUL - result)) {
 
        overflow = 1;
      }
      result += digit;
      anyDigits = 1;
    }
  }
 else if (base == 16) {
    unsigned long maxres = 0xFFFFFFFFUL >> 4;
    for ( ; ; p += 1) {
      digit = *p - 
'0'
;
      if (digit > (
'z'
 - 
'0'
)) {
        break;
      }
      digit = cvtIn[digit];
      if (digit > 15) {
        break;
      }
      if (result > maxres) {
 
        overflow = 1;
      }
      result = (result << 4);
      if (digit > (0xFFFFFFFFUL - result)) {
 
        overflow = 1;
      }
      result += digit;
      anyDigits = 1;
    }
  }
 else if ( base >= 2 && base <= 36 ) {
    unsigned long maxres = 0xFFFFFFFFUL / base;
    for ( ; ; p += 1) {
      digit = *p - 
'0'
;
      if (digit > (
'z'
 - 
'0'
)) {
        break;
      }
      digit = cvtIn[digit];
      if (digit >= ( (unsigned) base )) {
        break;
      }
      if (result > maxres) {
 
        overflow = 1;
      }
      result *= base;
      if (digit > (0xFFFFFFFFUL - result)) {
        overflow = 1;
      }
      result += digit;
      anyDigits = 1;
    }
  }
  
  if (!anyDigits) {
    p = string;
  }
  if (end_ptr != 0) {
    /* unsafe, but required by the strtoul prototype */
    *end_ptr = (char *) p;
  }
  if (overflow) {
    
    return 0xFFFFFFFFUL;
  }
 
  if (negative) {
    return -result;
  }
  return result;
}
/* Argument1: String of ASCII digits, possibly
 * preceded by white space.  For bases
 * greater than 10, either lower- or
 * upper-case digits may be used.
 */
/* Argument2: Where to store address of terminating
 * character, or NULL.
 */
/* Argument3: Base for conversion.  Must be less
 * than 37.  If 0, then the base is chosen
 * from the leading characters of string:
 * "0x" means hex, "0" means octal, anything
 * else means decimal.
 */
__device__
long int at_illecker_strtol(const char *string, char **end_ptr, int base) {
  register const char *p;
  long result;
  
  p = string;
  while (at_illecker_is_space((unsigned char) (*p))) {
    p += 1;
  }
  
  if (*p == 
'-'
) {
    p += 1;
    result = -(at_illecker_strtoul(p, end_ptr, base));
  }
 else {
    if (*p == 
'+'
) {
      p += 1;
    }
    result = at_illecker_strtoul(p, end_ptr, base);
  }
  if ((result == 0) && (end_ptr != 0) && (*end_ptr == p)) {
    *end_ptr = (char *) string;
  }
  return result;
}
__device__
double at_illecker_strtod(const char *string) {
  int sign = 0; 
  int expSign = 0; 
  double fraction, dblExp, *d;
  register const char *p;
  register int c;
  int exp = 0;
  int fracExp = 0;
  int mantSize;
  int decPt;
  const char *pExp;
  int maxExponent = 511;
  double powersOf10[] = {
    10.,
    100.,
    1.0e4,
    1.0e8,
    1.0e16,
    1.0e32,
    1.0e64,
    1.0e128,
    1.0e256
  }
;
  
  p = string;
  while (at_illecker_is_space((unsigned char) (*p))) {
    p += 1;
  }
  
  if (*p == 
'-'
) {
    sign = 1; 
    p += 1;
  }
 else {
    if (*p == 
'+'
) {
      p += 1;
    }
    sign = 0; 
  }
  
  
  decPt = -1;
  for (mantSize = 0; ; mantSize += 1) {
    c = *p;
    if (!at_illecker_is_digit(c)) {
      if ((c != 
'.'
) || (decPt >= 0)) {
        break;
      }
      decPt = mantSize;
    }
    p += 1;
  }
  
  
  
  
  pExp  = p;
  p -= mantSize;
  if (decPt < 0) {
    decPt = mantSize;
  }
 else {
    mantSize -= 1;
  }
  if (mantSize > 18) {
    fracExp = decPt - 18;
    mantSize = 18;
  }
 else {
    fracExp = decPt - mantSize;
  }
  if (mantSize == 0) {
    fraction = 0.0;
    p = string;
    goto done;
  }
 else {
    int frac1, frac2;
    frac1 = 0;
    for ( ; mantSize > 9; mantSize -= 1) {
      c = *p;
      p += 1;
      if (c == 
'.'
) {
        c = *p;
        p += 1;
      }
      frac1 = 10*frac1 + (c - 
'0'
);
    }
    frac2 = 0;
    for (; mantSize > 0; mantSize -= 1) {
      c = *p;
      p += 1;
      if (c == 
'.'
) {
        c = *p;
        p += 1;
      }
      frac2 = 10*frac2 + (c - 
'0'
);
    }
    fraction = (1.0e9 * frac1) + frac2;
  }
  
  p = pExp;
  if ((*p == 
'E'
) || (*p == 
'e'
)) {
    p += 1;
    if (*p == 
'-'
) {
      expSign = 1; 
      p += 1;
    }
 else {
      if (*p == 
'+'
) {
        p += 1;
      }
      expSign = 0; 
    }
    if (!at_illecker_is_digit((unsigned char) (*p))) {
      p = pExp;
      goto done;
    }
    while (at_illecker_is_digit((unsigned char) (*p))) {
      exp = exp * 10 + (*p - 
'0'
);
      p += 1;
    }
  }
  if (expSign) {
    exp = fracExp - exp;
  }
 else {
    exp = fracExp + exp;
  }
  
  
  
  
  if (exp < 0) {
    expSign = 1; 
    exp = -exp;
  }
 else {
    expSign = 0; 
  }
  if (exp > maxExponent) {
    exp = maxExponent;
    
    
  }
  dblExp = 1.0;
  for (d = powersOf10; exp != 0; exp >>= 1, d += 1) {
    if (exp & 01) {
      dblExp *= *d;
    }
  }
  if (expSign) {
    fraction /= dblExp;
  }
 else {
    fraction *= dblExp;
  }
done:
  if (sign) {
    return -fraction;
  }
  return fraction;
}
__device__
long java_lang_Long_parseLong(char * gc_info, int str_obj_ref, int * exception) {
  int str_value = 0;
  int str_count = 0;
  char str_val[255];
  long return_val = 0;
  str_value = instance_getter_java_lang_String_value(gc_info, str_obj_ref, exception);
  str_count = instance_getter_java_lang_String_count(gc_info, str_obj_ref, exception);
  
  
  for(int i = 0; i < str_count; i++){
    str_val[i] = char__array_get(gc_info, str_value, i, exception);
  }
  str_val[str_count] = 
'\0'
;
  
  return_val = at_illecker_strtol(str_val, 0, 0);
  
  return return_val;
}
__device__
int java_lang_Integer_parseInt(char * gc_info, int str_obj_ref, int * exception) {
  return java_lang_Long_parseLong(gc_info, str_obj_ref, exception);
}
__device__
double java_lang_Double_parseDouble(char * gc_info, int str_obj_ref, int * exception) {
  int str_value = 0;
  int str_count = 0;
  char str_val[255];
  double return_val = 0;
  str_value = instance_getter_java_lang_String_value(gc_info, str_obj_ref, exception);
  str_count = instance_getter_java_lang_String_count(gc_info, str_obj_ref, exception);
  
  
  for(int i = 0; i < str_count; i++){
    str_val[i] = char__array_get(gc_info, str_value, i, exception);
  }
  str_val[str_count] = 
'\0'
;
  
  return_val = at_illecker_strtod(str_val);
  
  return return_val;
}
/*****************************************************************************/
/* local typeof methods */
__device__ bool at_illecker_typeof_Integer(char * gc_info, int thisref){
  char * thisref_deref;
  GC_OBJ_TYPE_TYPE type;
  if(thisref == -1){
    return false;
  }
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  type = edu_syr_pcpratts_gc_get_type(thisref_deref);
  if(type==12647) {
    return true;
  }
  return false;
}
__device__ bool at_illecker_typeof_Long(char * gc_info, int thisref){
  char * thisref_deref;
  GC_OBJ_TYPE_TYPE type;
  if(thisref == -1){
    return false;
  }
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  type = edu_syr_pcpratts_gc_get_type(thisref_deref);
  if(type==12639) {
    return true;
  }
  return false;
}
__device__ bool at_illecker_typeof_Float(char * gc_info, int thisref){
  char * thisref_deref;
  GC_OBJ_TYPE_TYPE type;
  if(thisref == -1){
    return false;
  }
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  type = edu_syr_pcpratts_gc_get_type(thisref_deref);
  if(type==12643) {
    return true;
  }
  return false;
}
__device__ bool at_illecker_typeof_Double(char * gc_info, int thisref){
  char * thisref_deref;
  GC_OBJ_TYPE_TYPE type;
  if(thisref == -1){
    return false;
  }
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  type = edu_syr_pcpratts_gc_get_type(thisref_deref);
  if(type==12637) {
    return true;
  }
  return false;
}
__device__ bool at_illecker_typeof_String(char * gc_info, int thisref){
  char * thisref_deref;
  GC_OBJ_TYPE_TYPE type;
  if(thisref == -1){
    return false;
  }
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  type = edu_syr_pcpratts_gc_get_type(thisref_deref);
  if(type==2906) {
    return true;
  }
  return false;
}
/*****************************************************************************/
template<class T>
__device__
T at_illecker_getResult( char * gc_info, 
    HostDeviceInterface::MESSAGE_TYPE cmd, 
    HostDeviceInterface::TYPE return_type, bool use_return_value,
    int key_value_pair_ref, HostDeviceInterface::TYPE key_type, HostDeviceInterface::TYPE value_type,
    int int_param1, bool use_int_param1,
    int int_param2, bool use_int_param2,
    int int_param3, bool use_int_param3,
    long long long_param1, bool use_long_param1,
    long long long_param2, bool use_long_param2,
    float float_param1, bool use_float_param1,
    float float_param2, bool use_float_param2,
    double double_param1, bool use_double_param1,
    double double_param2, bool use_double_param2,
    int str_param1, bool use_str_param1,
    int str_param2, bool use_str_param2,
    int str_param3, bool use_str_param3,
    int * exception) {
  T return_value = 0;
  int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  int count = 0;
  int timeout = 0;
  bool done = false;
  int str_param1_value = 0;
  int str_param1_count = 0;
  int str_param2_value = 0;
  int str_param2_count = 0;
  int str_param3_value = 0;
  int str_param3_count = 0;
  int key_obj_ref = 0;
  int value_obj_ref = 0;
  char * key_obj_deref;
  char * value_obj_deref;
  
  while (count < 100) {
    
    if (++timeout > 100000) {
      break;
    }
    __syncthreads();
    
    if (done) {
      break;
    }
    
    int old = atomicCAS((int *) &host_device_interface->lock_thread_id, -1, thread_id);
    
    if (old == -1 || old == thread_id) {
      
      
      if (host_device_interface->is_debugging) {
        printf("gpu_Thread %d GOT LOCK lock_thread_id: %d\n", thread_id,
               host_device_interface->lock_thread_id);
      }
      /***********************************************************************/
      
      int inner_timeout = 0;
      while (host_device_interface->has_task) {
        
        if (++inner_timeout > 10000) {
          break;
        }
      }
      /***********************************************************************/
      
      host_device_interface->command = cmd;
      host_device_interface->return_type = return_type;
      
      if (use_int_param1) {
        host_device_interface->use_int_val1 = true;
        host_device_interface->int_val1 = int_param1;
      }
      if (use_int_param2) {
        host_device_interface->use_int_val2 = true;
        host_device_interface->int_val2 = int_param2;
      }
      if (use_int_param3) {
        host_device_interface->use_int_val3 = true;
        host_device_interface->int_val3 = int_param3;
      }
      if (use_long_param1) {
        host_device_interface->use_long_val1 = true;
        host_device_interface->long_val1 = long_param1;
      }
      if (use_long_param2) {
        host_device_interface->use_long_val2 = true;
        host_device_interface->long_val2 = long_param2;
      }
      if (use_float_param1) {
        host_device_interface->use_float_val1 = true;
        host_device_interface->float_val1 = float_param1;
      }
      if (use_float_param2) {
        host_device_interface->use_float_val2 = true;
        host_device_interface->float_val2 = float_param2;
      }
      if (use_double_param1) {
        host_device_interface->use_double_val1 = true;
        host_device_interface->double_val1 = double_param1;
      }
      if (use_double_param2) {
        host_device_interface->use_double_val2 = true;
        host_device_interface->double_val2 = double_param2;
      }
      if (use_str_param1) {
        str_param1_value = instance_getter_java_lang_String_value(gc_info, str_param1,
                          exception);
        str_param1_count = instance_getter_java_lang_String_count(gc_info, str_param1,
                          exception);
        
        for(int i = 0; i < str_param1_count; i++) {
          host_device_interface->str_val1[i] = char__array_get(gc_info, str_param1_value, i, exception);
        }
        host_device_interface->use_str_val1 = true;
        host_device_interface->str_val1[str_param1_count] = 
'\0'
;
      }
      if (use_str_param2) {
        str_param2_value = instance_getter_java_lang_String_value(gc_info, str_param2,
                           exception);
        str_param2_count = instance_getter_java_lang_String_count(gc_info, str_param2,
                           exception);
        
        for(int i = 0; i < str_param2_count; i++) {
          host_device_interface->str_val2[i] = char__array_get(gc_info, str_param2_value, i, exception);
        }
        host_device_interface->use_str_val2 = true;
        host_device_interface->str_val2[str_param2_count] = 
'\0'
;
      }
      if (use_str_param3) {
        str_param3_value = instance_getter_java_lang_String_value(gc_info, str_param3,
                           exception);
        str_param3_count = instance_getter_java_lang_String_count(gc_info, str_param3,
                           exception);
        
        for(int i = 0; i < str_param3_count; i++) {
          host_device_interface->str_val3[i] = char__array_get(gc_info, str_param3_value, i, exception);
        }
        host_device_interface->use_str_val3 = true;
        host_device_interface->str_val3[str_param3_count] = 
'\0'
;
      }
      
      host_device_interface->key_type = key_type;
      host_device_interface->value_type = value_type;
      /***********************************************************************/
      
      host_device_interface->has_task = true;
      __threadfence_system();
      
      /***********************************************************************/
      
      inner_timeout = 0;
      while (!host_device_interface->is_result_available) {
        __threadfence_system();
        
	
        if (++inner_timeout > 30000) {
          break;
        }
      }
      /***********************************************************************/
      
      if (return_type == HostDeviceInterface::KEY_VALUE_PAIR) {
        
        
        key_obj_ref = instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key(gc_info, 
                      key_value_pair_ref, exception);
        key_obj_deref = edu_syr_pcpratts_gc_deref(gc_info, key_obj_ref);
        
        if (key_type == HostDeviceInterface::INT) {
          *(( int *) &key_obj_deref[32]) = host_device_interface->int_val1;
        }
 else if (key_type == HostDeviceInterface::LONG) {
          *(( long long *) &key_obj_deref[32]) = host_device_interface->long_val1;
        }
 else if (key_type == HostDeviceInterface::FLOAT) {
          *(( float *) &key_obj_deref[32]) = host_device_interface->float_val1;
        }
 else if (key_type == HostDeviceInterface::DOUBLE) {
          *(( double *) &key_obj_deref[32]) = host_device_interface->double_val1;
        }
 else if (key_type == HostDeviceInterface::STRING) {
          int i;
          int len = at_illecker_strlen(host_device_interface->str_val1);
          int characters = char__array_new(gc_info, len, exception);
          for(i = 0; i < len; ++i) {
            char__array_set(gc_info, characters, i, host_device_interface->str_val1[i], exception);
          }
          
          *(( int *) &key_obj_deref[32]) = characters;
          
          *(( int *) &key_obj_deref[40]) = len;
          
          *(( int *) &key_obj_deref[44]) = 0;
        }
        
        value_obj_ref = instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value(gc_info, 
                        key_value_pair_ref, exception);
        value_obj_deref = edu_syr_pcpratts_gc_deref(gc_info, value_obj_ref);
        
        if (value_type == HostDeviceInterface::INT) {
          *(( int *) &value_obj_deref[32]) = host_device_interface->int_val2;
        }
 else if (value_type == HostDeviceInterface::LONG) {
          *(( long long *) &value_obj_deref[32]) = host_device_interface->long_val2;
        }
 else if (value_type == HostDeviceInterface::FLOAT) {
          *(( float *) &value_obj_deref[32]) = host_device_interface->float_val2;
        }
 else if (value_type == HostDeviceInterface::DOUBLE) {
          *(( double *) &value_obj_deref[32]) = host_device_interface->double_val2;
        }
 else if (value_type == HostDeviceInterface::STRING) {
          int i;
          int len = at_illecker_strlen(host_device_interface->str_val2);
          int characters = char__array_new(gc_info, len, exception);
          for(i = 0; i < len; ++i) {
            char__array_set(gc_info, characters, i, host_device_interface->str_val2[i], exception);
          }
          
          *(( int *) &value_obj_deref[32]) = characters;
          
          *(( int *) &value_obj_deref[40]) = len;
          
          *(( int *) &value_obj_deref[44]) = 0;
        }
        
        return_value = !host_device_interface->end_of_data;
      }
 else if (use_return_value) {
 
        
        if (return_type == HostDeviceInterface::INT) {
          return_value = host_device_interface->int_val1;
        }
 else if (return_type == HostDeviceInterface::LONG) {
          return_value = host_device_interface->long_val1;
        }
 else if (return_type == HostDeviceInterface::FLOAT) {
          return_value = host_device_interface->float_val1;
        }
 else if (return_type == HostDeviceInterface::DOUBLE) {
          return_value = host_device_interface->double_val1;
        }
 else if (return_type == HostDeviceInterface::STRING) {
          
          edu_syr_pcpratts_gc_assign(gc_info, (int*)&return_value,
            at_illecker_string_constant(gc_info, host_device_interface->str_val1, exception));
       
        }
 else if (return_type == HostDeviceInterface::STRING_ARRAY) {
          int index = 0;
          int array_len = host_device_interface->int_val1;
          if (array_len > 0) {
            
            return_value = java_lang_String__array_new(gc_info, array_len, exception);
            while ( (host_device_interface->use_int_val1) && (index < array_len) ) {
              if (host_device_interface->use_str_val1) {
                java_lang_String__array_set(gc_info, return_value, index, 
                  at_illecker_string_constant(gc_info, host_device_interface->str_val1, exception), exception);
                index++;
              }
              if (host_device_interface->use_str_val2) {
                java_lang_String__array_set(gc_info, return_value, index, 
                  at_illecker_string_constant(gc_info, host_device_interface->str_val2, exception), exception);
                index++;
              }
              if (host_device_interface->use_str_val3) {
                java_lang_String__array_set(gc_info, return_value, index, 
                  at_illecker_string_constant(gc_info, host_device_interface->str_val3, exception), exception);
                index++;
              }
              
              host_device_interface->is_result_available = false;
              __threadfence_system();
              
              while (!host_device_interface->is_result_available) {
                __threadfence_system();
              }
            }
          }
 else {
            return_value = 0;
          }
        }
      }
      /***********************************************************************/
      
      if ( (use_int_param1) || (return_type == HostDeviceInterface::INT) ) {
        host_device_interface->int_val1 = 0;
        host_device_interface->use_int_val1 = false;
      }
      if (use_int_param2) {
        host_device_interface->int_val2 = 0;
        host_device_interface->use_int_val2 = false;
      }
      if (use_int_param3) {
        host_device_interface->int_val3 = 0;
        host_device_interface->use_int_val3 = false;
      }
      if ( (use_long_param1) || (return_type == HostDeviceInterface::LONG) ) {
        host_device_interface->long_val1 = 0;
        host_device_interface->use_long_val1 = false;
      }
      if (use_long_param1) {
        host_device_interface->long_val2 = 0;
        host_device_interface->use_long_val2 = false;
      }
      if ( (use_float_param1) || (return_type == HostDeviceInterface::FLOAT) ) {
        host_device_interface->float_val1 = 0;
        host_device_interface->use_float_val1 = false;
      }
      if (use_float_param2) {
        host_device_interface->float_val2 = 0;
        host_device_interface->use_float_val2 = false;
      }
      if ( (use_double_param1) || (return_type == HostDeviceInterface::DOUBLE) ) {
        host_device_interface->double_val1 = 0;
        host_device_interface->use_double_val1 = false;
      }
      if (use_double_param2) {
        host_device_interface->double_val2 = 0;
        host_device_interface->use_double_val2 = false;
      }
      if ( (use_str_param1) || (return_type == HostDeviceInterface::STRING) ) {
        host_device_interface->str_val1[0] = 
'\0'
;
        host_device_interface->use_str_val1 = false;
      }
      if (use_str_param2) {
        host_device_interface->str_val2[0] = 
'\0'
;
        host_device_interface->use_str_val2 = false;
      }
      if (use_str_param3) {
        host_device_interface->str_val3[0] = 
'\0'
;
        host_device_interface->use_str_val3 = false;
      }
      if (return_type == HostDeviceInterface::STRING_ARRAY) {
        host_device_interface->int_val1 = 0;
        host_device_interface->use_int_val1 = false;
        host_device_interface->str_val1[0] = 
'\0'
;
        host_device_interface->use_str_val1 = false;
        host_device_interface->str_val2[0] = 
'\0'
;
        host_device_interface->use_str_val2 = false;
        host_device_interface->str_val3[0] = 
'\0'
;
        host_device_interface->use_str_val3 = false;
      }
      host_device_interface->command = HostDeviceInterface::UNDEFINED;
      host_device_interface->return_type = HostDeviceInterface::NOT_AVAILABLE;
      host_device_interface->key_type = HostDeviceInterface::NOT_AVAILABLE;
      host_device_interface->value_type = HostDeviceInterface::NOT_AVAILABLE;
      /***********************************************************************/ 
      
      host_device_interface->is_result_available = false;
      host_device_interface->lock_thread_id = -1;
      
      __threadfence_system();
      
      /***********************************************************************/ 
      
      done = true; 
    }
 else {
      count++;
      if (count > 50) {
        count = 0;
      }
    }
  }
  return return_value;
}
/*****************************************************************************/
/* Hama Peer public methods */
__device__
void edu_syr_pcpratts_rootbeer_runtime_HamaPeer_send( char * gc_info,
     int peer_name_str_ref, int message_obj_ref, int * exception) {
  int int_value = 0;
  bool use_int_value = false;
  long long long_value = 0;
  bool use_long_value = false;
  float float_value = 0;
  bool use_float_value = false;
  double double_value = 0;
  bool use_double_value = false;
  int string_value = 0;
  bool use_string_value = false;
  char * message_obj_deref;
  
  
  if (message_obj_ref == -1) {
    printf("Exception in HamaPeer.send: unsupported NULL Type\n");
    return;
  }
 else {
    
    if (at_illecker_typeof_Integer(gc_info, message_obj_ref)) {
      message_obj_deref = edu_syr_pcpratts_gc_deref(gc_info, message_obj_ref);
      int_value = *(( int *) &message_obj_deref[32]);
      use_int_value = true;
      
    }
 else if (at_illecker_typeof_Long(gc_info, message_obj_ref)) {
      message_obj_deref = edu_syr_pcpratts_gc_deref(gc_info, message_obj_ref);
      long_value = *(( long long *) &message_obj_deref[32]);
      use_long_value = true;
      
    }
 else if (at_illecker_typeof_Float(gc_info, message_obj_ref)) {
      message_obj_deref = edu_syr_pcpratts_gc_deref(gc_info, message_obj_ref);
      float_value = *(( float *) &message_obj_deref[32]);
      use_float_value = true;
      
    }
 else if (at_illecker_typeof_Double(gc_info, message_obj_ref)) {
      message_obj_deref = edu_syr_pcpratts_gc_deref(gc_info, message_obj_ref);
      double_value = *(( double *) &message_obj_deref[32]);
      use_double_value = true;
      
    }
 else if (at_illecker_typeof_String(gc_info, message_obj_ref)) {
      string_value = message_obj_ref;
      use_string_value = true;
      
    }
 else {
      
      printf("Exception in HamaPeer.send: unsupported Type\n");
      return;
    }
  }
  at_illecker_getResult<int>(gc_info, HostDeviceInterface::SEND_MSG,
    HostDeviceInterface::NOT_AVAILABLE, false, 
    0, HostDeviceInterface::NOT_AVAILABLE, HostDeviceInterface::NOT_AVAILABLE,
    int_value, use_int_value,
    0, false,
    0, false,
    long_value, use_long_value,
    0, false,
    float_value, use_float_value,
    0, false,
    double_value, use_double_value,
    0, false,
    peer_name_str_ref, true,
    string_value, use_string_value,
    0, false,
    exception);
}
__device__
int edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getCurrentStringMessage( char * gc_info, 
    int * exception) {
  return at_illecker_getResult<int>(gc_info, HostDeviceInterface::GET_MSG,
           HostDeviceInterface::STRING, true, 
           0, HostDeviceInterface::NOT_AVAILABLE, HostDeviceInterface::NOT_AVAILABLE,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           exception);
}
__device__
int edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getNumCurrentMessages( char * gc_info, 
    int * exception) {
  return at_illecker_getResult<int>(gc_info, HostDeviceInterface::GET_MSG_COUNT,
           HostDeviceInterface::INT, true, 
           0, HostDeviceInterface::NOT_AVAILABLE, HostDeviceInterface::NOT_AVAILABLE,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           exception);
}
__device__
void edu_syr_pcpratts_rootbeer_runtime_HamaPeer_sync( char * gc_info, 
     int * exception) {
  at_illecker_getResult<int>(gc_info, HostDeviceInterface::SYNC,
    HostDeviceInterface::NOT_AVAILABLE, false, 
    0, HostDeviceInterface::NOT_AVAILABLE, HostDeviceInterface::NOT_AVAILABLE,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    exception);
}
__device__
long edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getSuperstepCount( char * gc_info, 
    int * exception) {
  return at_illecker_getResult<long>(gc_info, HostDeviceInterface::GET_SUPERSTEP_COUNT,
           HostDeviceInterface::LONG, true, 
           0, HostDeviceInterface::NOT_AVAILABLE, HostDeviceInterface::NOT_AVAILABLE,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           exception);
}
__device__
int edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getAllPeerNames( char * gc_info, 
    int * exception) {
  return at_illecker_getResult<int>(gc_info, HostDeviceInterface::GET_ALL_PEERNAME,
           HostDeviceInterface::STRING_ARRAY, true, 
           0, HostDeviceInterface::NOT_AVAILABLE, HostDeviceInterface::NOT_AVAILABLE,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           exception);
}
__device__
bool edu_syr_pcpratts_rootbeer_runtime_HamaPeer_readNext( char * gc_info, 
     int key_value_pair_ref, int * exception) {
  int key_obj_ref;
  int value_obj_ref;
  HostDeviceInterface::TYPE key_type;
  HostDeviceInterface::TYPE value_type;
  key_obj_ref = instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key(gc_info, 
                key_value_pair_ref, exception);
  value_obj_ref = instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value(gc_info, 
                  key_value_pair_ref, exception);
  
  if (at_illecker_typeof_Integer(gc_info, key_obj_ref)) {
    key_type = HostDeviceInterface::INT;
  }
 else if (at_illecker_typeof_Long(gc_info, key_obj_ref)) {
    key_type = HostDeviceInterface::LONG;
  }
 else if (at_illecker_typeof_Float(gc_info, key_obj_ref)) {
    key_type = HostDeviceInterface::FLOAT;
  }
 else if (at_illecker_typeof_Double(gc_info, key_obj_ref)) {
    key_type = HostDeviceInterface::DOUBLE;
  }
 else if (at_illecker_typeof_String(gc_info, key_obj_ref)) {
    key_type = HostDeviceInterface::STRING;
  }
 else if (key_obj_ref == -1) {
    key_type = HostDeviceInterface::NULL_TYPE;
  }
 else {
    
    printf("Exception in HamaPeer.readNext: unsupported Key Type\n");
    return false;
  }
  
  if (at_illecker_typeof_Integer(gc_info, value_obj_ref)) {
    value_type = HostDeviceInterface::INT;
  }
 else if (at_illecker_typeof_Long(gc_info, value_obj_ref)) {
    value_type = HostDeviceInterface::LONG;
  }
 else if (at_illecker_typeof_Float(gc_info, value_obj_ref)) {
    value_type = HostDeviceInterface::FLOAT;
  }
 else if (at_illecker_typeof_Double(gc_info, value_obj_ref)) {
    value_type = HostDeviceInterface::DOUBLE;
  }
 else if (at_illecker_typeof_String(gc_info, value_obj_ref)) {
    value_type = HostDeviceInterface::STRING;
  }
 else if (value_obj_ref == -1) {
    value_type = HostDeviceInterface::NULL_TYPE;
  }
 else {
    
    printf("Exception in HamaPeer.readNext: unsupported Value Type\n");
    return false;
  }
  if ( (key_type == HostDeviceInterface::NULL_TYPE) &&
       (value_type == HostDeviceInterface::NULL_TYPE) ) {
    printf("Exception in HamaPeer.readNext: key and value are NULL!\n");
    return false;
  }
  return at_illecker_getResult<int>(gc_info, HostDeviceInterface::READ_KEYVALUE,
           HostDeviceInterface::KEY_VALUE_PAIR, false, 
           key_value_pair_ref, key_type, value_type,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           exception);
}
__device__ int java_lang_String__array_get( char * gc_info, int thisref, int parameter0, int * exception){
int offset;
int length;
 char * thisref_deref;
offset = 32+(parameter0*4);
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return 0;
}
return *(( int *) &thisref_deref[offset]);
}
__device__ void java_lang_String__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception){
int length;
 char * thisref_deref;
  if(thisref == -1){
    *exception = 21352;
    return;
  }
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return;
}
*(( int *) &thisref_deref[32+(parameter0*4)]) = parameter1;
}
__device__ int java_lang_String__array_new( char * gc_info, int size, int * exception){
int i;
int total_size;
int mod;
int thisref;
 char * thisref_deref;
total_size = (size * 4)+ 32;
mod = total_size % 8;
if(mod != 0)
  total_size += (8 - mod);
thisref = edu_syr_pcpratts_gc_malloc(gc_info, total_size);
if(thisref == -1){
  *exception = 21352;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 2902);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, total_size);
edu_syr_pcpratts_setint(thisref_deref, 12, size);
for(i = 0; i < size; ++i){
  java_lang_String__array_set(gc_info, thisref, i, -1, exception);
}
return thisref;
}
__device__ int edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_getKey13_( char * gc_info, int thisref, int * exception){
int r0 = -1;
int $r1 = -1;
 r0  =  thisref ;
 $r1  = instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
return  $r1 ;
  return 0;
}
__device__ int int__array_get( char * gc_info, int thisref, int parameter0, int * exception){
int offset;
int length;
 char * thisref_deref;
offset = 32+(parameter0*4);
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return 0;
}
return *(( int *) &thisref_deref[offset]);
}
__device__ int java_lang_String_initab850b60f96d11de8a390800200c9a660_a14_5_5_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception){
int r0 = -1;
int r1 = -1;
int i0;
int i1;
int $r2 = -1;
int $r3 = -1;
int $i2;
int $i3;
int $r4 = -1;
int $i4;
int $i5;
int $r5 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 64));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 1);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 2906);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 64);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_java_lang_String_value(gc_info, thisref, -1, exception);
instance_setter_java_lang_String_count(gc_info, thisref, 0, exception);
instance_setter_java_lang_String_hash(gc_info, thisref, 0, exception);
instance_setter_java_lang_String_offset(gc_info, thisref, 0, exception);
 r0  =  thisref ;
 r1  =  parameter0 ;
 i0  =  parameter1 ;
 i1  =  parameter2 ;
if ( i0  >=  0   ) goto label0;
 $r2  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r2, java_lang_StringIndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a660_5_(gc_info,
  i0 , exception));
if(*exception != 0) {
 
return 0; }
 *exception =  $r2 ;
return 0;
label0:
if ( i1  >=  0   ) goto label1;
 $r3  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r3, java_lang_StringIndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a660_5_(gc_info,
  i1 , exception));
if(*exception != 0) {
 
return 0; }
 *exception =  $r3 ;
return 0;
label1:
 $i2  = edu_syr_pcpratts_array_length(gc_info,  r1 );
 $i3  =  $i2  -  i1  ;
if ( i0  <=  $i3   ) goto label2;
 $r4  =  -1 ;
 $i4  =  i0  +  i1  ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r4, java_lang_StringIndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a660_5_(gc_info,
  $i4 , exception));
if(*exception != 0) {
 
return 0; }
 *exception =  $r4 ;
return 0;
label2:
instance_setter_java_lang_String_offset(gc_info, r0,  0 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_java_lang_String_count(gc_info, r0,  i1 , exception);
if(*exception != 0) {
 
return 0; }
 $i5  =  i0  +  i1  ;
 $r5  = java_util_Arrays_copyOfRangea14_a14_5_5_(gc_info,  r1 ,  i0 ,  $i5 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_java_lang_String_value(gc_info, r0,  $r5 , exception);
if(*exception != 0) {
 
return 0; }
return r0;
  return 0;
}
__device__ void java_lang_IndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
 r0  =  thisref ;
 r1  =  parameter0 ;
java_lang_RuntimeException_initab850b60f96d11de8a390800200c9a66_body0_9_(gc_info,
 thisref,
  r1 , exception);
return;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_toArraya12_( char * gc_info, int thisref, int * exception){
int r0 = -1;
int r1 = -1;
int i0;
int $i1;
int $i2;
int $r2 = -1;
double $d0;
 r0  =  thisref ;
 $i1  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 r1  = double__array_new(gc_info,  $i1 , exception);
 i0  =  0 ;
label1:
 $i2  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
if ( i0  >=  $i2   ) goto label0;
 $r2  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 $d0  = double__array_get(gc_info, $r2, i0, exception);
if(*exception != 0) {
 
return 0; }
double__array_set(gc_info, r1, i0,  $d0 , exception);
if(*exception != 0) {
 
return 0; }
 i0  =  i0  +  1  ;
goto label1;
label0:
return  r1 ;
  return 0;
}
__device__ void java_lang_RuntimeException_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
 r0  =  thisref ;
 r1  =  parameter0 ;
java_lang_Exception_initab850b60f96d11de8a390800200c9a66_body0_9_(gc_info,
 thisref,
  r1 , exception);
return;
}
__device__ void at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_assignCenters0_5_5_5_5_5_( char * gc_info, int thisref, int parameter0, int parameter1, int parameter2, int parameter3, int parameter4, int * exception){
int r0 = -1;
int i0;
int i1;
int i2;
int i3;
int i4;
int i5;
int i6;
int i7;
int i8;
int i9;
int $i10;
int $i11;
int $i12;
int $i13;
int $i14;
int $i15;
int $i16;
int $i17;
double $d0;
int i18;
int $i19;
int $i20;
int $i21;
int i22;
int $i23;
int $i24;
int $i25;
int $i26;
int i27;
double $d1;
double $d2;
double $d3;
int $i28;
int id;
char * mem;
char * trash;
char * mystery;
int count;
int old;
char * thisref_synch_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
id = getThreadId();
mem = edu_syr_pcpratts_gc_deref(gc_info, thisref);
trash = edu_syr_pcpratts_gc_deref(gc_info, 0) + 220;
mystery = trash - 8;
mem += 16;
count = 0;
while(count < 100){
  old = atomicCAS((int *) mem, -1 , id);
  *((int *) trash) = old;
  if(old == -1 || old == id){
  if ( thisref ==-1 ) {
 
    * exception = 11;
  }
  if ( * exception != 0 ) {
    edu_syr_pcpratts_exitMonitorMem ( gc_info , mem , old ) ;
    return;
  }
  thisref_synch_deref = edu_syr_pcpratts_gc_deref ( gc_info , thisref );
  * ( ( int * ) & thisref_synch_deref [ 20 ] ) = 20 ;
 r0  =  thisref ;
 i0  =  parameter0 ;
 i1  =  parameter1 ;
 i2  =  parameter2 ;
 i3  =  parameter3 ;
 i4  =  parameter4 ;
 $i10  =  i0  *  4  ;
 i5  =  i3  +  $i10  ;
 i6  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedInteger(gc_info,  i5 , exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
if ( i6  !=  -1   ) goto label0;
 i7  =  0 ;
label2:
if ( i7  >=  i1   ) goto label1;
 $i11  =  i0  *  i1  ;
 $i12  =  $i11  +  i7  ;
 $i13  =  $i12  *  8  ;
 i8  =  i2  +  $i13  ;
 $i14  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getThreadIdxx(gc_info, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $i15  =  $i14  *  i1  ;
 $i16  =  $i15  +  i7  ;
 $i17  =  $i16  *  8  ;
 i9  =  i4  +  $i17  ;
 $d0  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedDouble(gc_info,  i9 , exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedDouble(gc_info,  i8 ,  $d0 , exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 i7  =  i7  +  1  ;
goto label2;
label1:
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedInteger(gc_info,  i5 ,  0 , exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
goto label3;
label0:
 i18  =  0 ;
label5:
if ( i18  >=  i1   ) goto label4;
 $i19  =  i0  *  i1  ;
 $i20  =  $i19  +  i18  ;
 $i21  =  $i20  *  8  ;
 i22  =  i2  +  $i21  ;
 $i23  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getThreadIdxx(gc_info, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $i24  =  $i23  *  i1  ;
 $i25  =  $i24  +  i18  ;
 $i26  =  $i25  *  8  ;
 i27  =  i4  +  $i26  ;
 $d1  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedDouble(gc_info,  i22 , exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $d2  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedDouble(gc_info,  i27 , exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $d3  =  $d1  +  $d2  ;
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedDouble(gc_info,  i22 ,  $d3 , exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 i18  =  i18  +  1  ;
goto label5;
label4:
 $i28  =  i6  +  1  ;
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedInteger(gc_info,  i5 ,  $i28 , exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
label3:
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return;
  }
 else {
    count++;
    if(count > 50 || (*((int *) mystery)) == 0){
      count = 0;
    }
  }
}
}
__device__ int java_lang_Integer_toString9_5_( char * gc_info, int parameter0, int * exception){
int i0;
int i1;
int r0 = -1;
int $i2;
int $i3;
int $i4;
int $r1 = -1;
 i0  =  parameter0 ;
if ( i0  !=  -2147483648   ) goto label0;
return  edu_syr_pcpratts_string_constant(gc_info, (char *) "-2147483648", exception) ;
label0:
if ( i0  >=  0   ) goto label1;
 $i2  = - i0 ;
 $i3  = java_lang_Integer_stringSize5_5_(gc_info,  $i2 , exception);
if(*exception != 0) {
 
return 0; }
 $i4  =  $i3  +  1  ;
goto label2;
label1:
 $i4  = java_lang_Integer_stringSize5_5_(gc_info,  i0 , exception);
if(*exception != 0) {
 
return 0; }
label2:
 i1  =  $i4 ;
 r0  = char__array_new(gc_info,  i1 , exception);
java_lang_Integer_getChars0_5_5_a14_(gc_info,  i0 ,  i1 ,  r0 , exception);
if(*exception != 0) {
 
return 0; }
 $r1  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r1, java_lang_String_initab850b60f96d11de8a390800200c9a660_5_5_a14_(gc_info,
  0 ,
  i1 ,
  r0 , exception));
if(*exception != 0) {
 
return 0; }
return  $r1 ;
  return 0;
}
__device__ int java_lang_Integer_toUnsignedString9_5_5_( char * gc_info, int parameter0, int parameter1, int * exception){
int i0;
int i1;
int r0 = -1;
int i2;
int i3;
int i4;
int $r1 = -1;
int $i5;
char $c6;
int $r2 = -1;
int $i7;
 i0  =  parameter0 ;
 i1  =  parameter1 ;
 r0  = char__array_new(gc_info,  32 , exception);
 i2  =  32 ;
 i3  =  1  <<  i1  ;
 i4  =  i3  -  1  ;
label0:
 i2  =  i2  +  -1  ;
 $r1  = static_getter_java_lang_Integer_digits(gc_info, exception);
 $i5  =  i0  &  i4  ;
 $c6  = char__array_get(gc_info, $r1, $i5, exception);
if(*exception != 0) {
 
return 0; }
char__array_set(gc_info, r0, i2,  $c6 , exception);
if(*exception != 0) {
 
return 0; }
 i0  = ( i0  >>  i1  ) & 0x7fffffff;
if ( i0  !=  0   ) goto label0;
 $r2  =  -1 ;
 $i7  =  32  -  i2  ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r2, java_lang_String_initab850b60f96d11de8a390800200c9a660_a14_5_5_(gc_info,
  r0 ,
  i2 ,
  $i7 , exception));
if(*exception != 0) {
 
return 0; }
return  $r2 ;
  return 0;
}
__device__ int java_lang_String_initab850b60f96d11de8a390800200c9a660_5_5_a14_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception){
int r0 = -1;
int i0;
int i1;
int r1 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 64));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 1);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 2906);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 64);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_java_lang_String_value(gc_info, thisref, -1, exception);
instance_setter_java_lang_String_count(gc_info, thisref, 0, exception);
instance_setter_java_lang_String_hash(gc_info, thisref, 0, exception);
instance_setter_java_lang_String_offset(gc_info, thisref, 0, exception);
 r0  =  thisref ;
 i0  =  parameter0 ;
 i1  =  parameter1 ;
 r1  =  parameter2 ;
instance_setter_java_lang_String_value(gc_info, r0,  r1 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_java_lang_String_offset(gc_info, r0,  i0 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_java_lang_String_count(gc_info, r0,  i1 , exception);
if(*exception != 0) {
 
return 0; }
return r0;
  return 0;
}
__device__ double java_lang_Math_abs8_8_( char * gc_info, double parameter0, int * exception){
double d0;
char $b0;
double $d1;
 d0  =  parameter0 ;
 $b0  = edu_syr_pcpratts_cmpg((double) d0 , (double) 0.0 );
if ( $b0  >  0   ) goto label0;
 $d1  =  0.0  -  d0  ;
goto label1;
label0:
 $d1  =  d0 ;
label1:
return  $d1 ;
  return 0;
}
__device__ void java_lang_Exception_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
 r0  =  thisref ;
 r1  =  parameter0 ;
java_lang_Throwable_initab850b60f96d11de8a390800200c9a66_body0_9_(gc_info,
 thisref,
  r1 , exception);
return;
}
__device__ char char__array_get( char * gc_info, int thisref, int parameter0, int * exception){
int offset;
int length;
 char * thisref_deref;
offset = 32+(parameter0*4);
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return 0;
}
return *(( char *) &thisref_deref[offset]);
}
__device__ void char__array_set( char * gc_info, int thisref, int parameter0, char parameter1, int * exception){
int length;
 char * thisref_deref;
  if(thisref == -1){
    *exception = 21352;
    return;
  }
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return;
}
*(( int *) &thisref_deref[32+(parameter0*4)]) = 0;
*(( char *) &thisref_deref[32+(parameter0*4)]) = parameter1;
}
__device__ int char__array_new( char * gc_info, int size, int * exception){
int i;
int total_size;
int mod;
int thisref;
 char * thisref_deref;
total_size = (size * 4)+ 32;
mod = total_size % 8;
if(mod != 0)
  total_size += (8 - mod);
thisref = edu_syr_pcpratts_gc_malloc(gc_info, total_size);
if(thisref == -1){
  *exception = 21352;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 4335);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, total_size);
edu_syr_pcpratts_setint(thisref_deref, 12, size);
for(i = 0; i < size; ++i){
  char__array_set(gc_info, thisref, i, 0, exception);
}
return thisref;
}
__device__ int edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_initab850b60f96d11de8a390800200c9a660_13_13_( char * gc_info, int parameter0, int parameter1, int * exception){
int r0 = -1;
int r1 = -1;
int r2 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 48));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 2);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 3421);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 48);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key(gc_info, thisref, -1, exception);
instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value(gc_info, thisref, -1, exception);
 r0  =  thisref ;
 r1  =  parameter0 ;
 r2  =  parameter1 ;
instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key(gc_info, r0,  r1 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value(gc_info, r0,  r2 , exception);
if(*exception != 0) {
 
return 0; }
return r0;
  return 0;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception){
int r0 = -1;
int $r1 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 48));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 1);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 2911);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 48);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, thisref, -1, exception);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, thisref, 0, exception);
 r0  =  thisref ;
 $r1  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_new(gc_info,  8 , exception);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, r0,  $r1 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0,  0 , exception);
if(*exception != 0) {
 
return 0; }
return r0;
  return 0;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_getLength5_( char * gc_info, int thisref, int * exception){
int r0 = -1;
int $i0;
 r0  =  thisref ;
 $i0  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
return  $i0 ;
  return 0;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_get( char * gc_info, int thisref, int parameter0, int * exception){
int offset;
int length;
 char * thisref_deref;
offset = 32+(parameter0*4);
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return 0;
}
return *(( int *) &thisref_deref[offset]);
}
__device__ void at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception){
int length;
 char * thisref_deref;
  if(thisref == -1){
    *exception = 21352;
    return;
  }
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return;
}
*(( int *) &thisref_deref[32+(parameter0*4)]) = parameter1;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_new( char * gc_info, int size, int * exception){
int i;
int total_size;
int mod;
int thisref;
 char * thisref_deref;
total_size = (size * 4)+ 32;
mod = total_size % 8;
if(mod != 0)
  total_size += (8 - mod);
thisref = edu_syr_pcpratts_gc_malloc(gc_info, total_size);
if(thisref == -1){
  *exception = 21352;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 11550);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, total_size);
edu_syr_pcpratts_setint(thisref_deref, 12, size);
for(i = 0; i < size; ++i){
  at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_set(gc_info, thisref, i, -1, exception);
}
return thisref;
}
__device__ int double__array__array_get( char * gc_info, int thisref, int parameter0, int * exception){
int offset;
int length;
 char * thisref_deref;
offset = 32+(parameter0*4);
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return 0;
}
return *(( int *) &thisref_deref[offset]);
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_getNearestCenter5_5_5_5_( char * gc_info, int thisref, int parameter0, int parameter1, int parameter2, int * exception){
int r0 = -1;
int i0;
int i1;
int i2;
int i3;
double d0;
int i4;
double d1;
char $b5;
 r0  =  thisref ;
 i0  =  parameter0 ;
 i1  =  parameter1 ;
 i2  =  parameter2 ;
 i3  =  0 ;
 d0  =  1.7976931348623157E308 ;
 i4  =  0 ;
label2:
if ( i4  >=  i0   ) goto label0;
 d1  = at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_measureEuclidianDistance8_5_5_5_(gc_info,
 r0,
  i4 ,
  i1 ,
  i2 , exception);
if(*exception != 0) {
 
return 0; }
 $b5  = edu_syr_pcpratts_cmpg((double) d1 , (double) d0 );
if ( $b5  >=  0   ) goto label1;
 d0  =  d1 ;
 i3  =  i4 ;
label1:
 i4  =  i4  +  1  ;
goto label2;
label0:
return  i3 ;
  return 0;
}
__device__ void 
java_lang_System_arraycopy( char * gc_info, int src_handle, int srcPos, int dest_handle, int destPos, int length, int * exception){
  int i;
  int src_index;
  int dest_index;
   char * src_deref = edu_syr_pcpratts_gc_deref(gc_info, src_handle);
   char * dest_deref = edu_syr_pcpratts_gc_deref(gc_info, dest_handle);
  
  GC_OBJ_TYPE_TYPE src_type = edu_syr_pcpratts_gc_get_type(src_deref);
  GC_OBJ_TYPE_TYPE dest_type = edu_syr_pcpratts_gc_get_type(dest_deref);
  
  if(srcPos < destPos){
      if(0){
}
      else if(src_type == 4335 && dest_type == 4335){
        for(i = length - 1; i >= 0; --i){
          src_index = srcPos + i;
          dest_index = destPos + i;
        char__array_set(gc_info, dest_handle, dest_index, char__array_get(gc_info, src_handle, src_index, exception), exception);
        }
      }
  }
 else {
      if(0){
}
      else if(src_type == 4335 && dest_type == 4335){
        for(i = length - 1; i >= 0; --i){
          src_index = srcPos + i;
          dest_index = destPos + i;
        char__array_set(gc_info, dest_handle, dest_index, char__array_get(gc_info, src_handle, src_index, exception), exception);
        }
      }
  }
}
__device__ int invoke_java_lang_Object_hashCode( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
GC_OBJ_TYPE_TYPE derived_type;
if(thisref == -1){
  *exception = -2;
return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
derived_type = edu_syr_pcpratts_gc_get_type(thisref_deref);
if(0){
}
else if(derived_type == 1){
return java_lang_Object_hashCode(gc_info, thisref, exception);
}
else if(derived_type == 2906){
return java_lang_String_hashCode5_(gc_info, thisref, exception);
}
return -1;
}
__device__ double at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_measureEuclidianDistance8_5_5_5_( char * gc_info, int thisref, int parameter0, int parameter1, int parameter2, int * exception){
int r0 = -1;
int i0;
int i1;
int i2;
double d0;
int i3;
int i4;
int i5;
double d1;
int $i6;
int $i7;
int $i8;
int $i9;
int $i10;
int $i11;
double $d2;
double $d3;
double $d4;
double $d5;
 r0  =  thisref ;
 i0  =  parameter0 ;
 i1  =  parameter1 ;
 i2  =  parameter2 ;
 d0  =  0.0 ;
 i3  =  0 ;
label1:
if ( i3  >=  i1   ) goto label0;
 $i6  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getThreadIdxx(gc_info, exception);
if(*exception != 0) {
 
return 0; }
 $i7  =  $i6  *  i1  ;
 $i8  =  $i7  +  i3  ;
 $i9  =  $i8  *  8  ;
 i4  =  i2  +  $i9  ;
 $i10  =  i0  *  i1  ;
 $i11  =  $i10  +  i3  ;
 i5  =  $i11  *  8  ;
 $d2  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedDouble(gc_info,  i4 , exception);
if(*exception != 0) {
 
return 0; }
 $d3  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedDouble(gc_info,  i5 , exception);
if(*exception != 0) {
 
return 0; }
 d1  =  $d2  -  $d3  ;
 $d4  =  d1  *  d1  ;
 d0  =  d0  +  $d4  ;
 i3  =  i3  +  1  ;
goto label1;
label0:
 $d5  = java_lang_Math_sqrt8_8_(gc_info,  d0 , exception);
if(*exception != 0) {
 
return 0; }
return  $d5 ;
  return 0;
}
__device__ void at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_gpuMethod0_( char * gc_info, int thisref, int * exception){
int r0 = -1;
int i0;
int i1;
int i2;
int i3;
int i4;
int i5;
int i6;
int i7;
int i8;
int i9;
int i10;
int i11;
int i12;
int r1 = -1;
int r2 = -1;
int r3 = -1;
int r4 = -1;
int i13;
int i14;
int i15;
int r5 = -1;
int i16;
int i17;
int i18;
int i19;
int $r6 = -1;
int $r7 = -1;
int $r8 = -1;
int $i20;
int $i21;
int $i22;
int $i23;
int $i24;
int $i25;
int $r9 = -1;
int $r10 = -1;
int $r11 = -1;
int $r12 = -1;
int $i26;
int $i27;
int $r13 = -1;
int $r14 = -1;
double $d0;
int $i28;
int $i29;
int $i30;
int $i31;
int i32;
char z0;
char z1;
int i33;
int i34;
int $r15 = -1;
int $r16 = -1;
int $r17 = -1;
int $r18 = -1;
int $r19 = -1;
int r20 = -1;
int $r21 = -1;
int $r22 = -1;
int $i35;
int $i36;
int $i37;
double $d1;
int i38;
int $r23 = -1;
int r24 = -1;
int r25 = -1;
int i39;
int $i40;
int $i41;
int $i42;
int i43;
double $d2;
int $r26 = -1;
int $i44;
int i45;
int r27 = -1;
int i46;
int $i47;
int i48;
int i49;
int r28 = -1;
int $r29 = -1;
int $r30 = -1;
int $r31 = -1;
int $r32 = -1;
int r33 = -1;
int $r34 = -1;
int $r35 = -1;
int $r36 = -1;
int r37 = -1;
int $r38 = -1;
int $r39 = -1;
int $r40 = -1;
int $r41 = -1;
int r42 = -1;
int $r43 = -1;
int $r44 = -1;
int $r45 = -1;
int r46 = -1;
int i50;
int $i51;
int $i52;
int $i53;
int i54;
int $r47 = -1;
int $r48 = -1;
double $d3;
int $r49 = -1;
int $r50 = -1;
int $i55;
int $r51 = -1;
int $r52 = -1;
int $r53 = -1;
int $r54 = -1;
int $r55 = -1;
int $r56 = -1;
int r57 = -1;
int i56;
int i57;
int i58;
int $i59;
int $i60;
int $i61;
int i62;
int $i63;
int i64;
int i65;
int i66;
int r58 = -1;
int $r59 = -1;
int $r60 = -1;
int $r61 = -1;
int r62 = -1;
int $r63 = -1;
int i67;
int $r64 = -1;
int i68;
int $r65 = -1;
int r66 = -1;
int i69;
int r67 = -1;
int $r68 = -1;
double $d4;
int $i70;
int i71;
int $i72;
int $i73;
int $i74;
double $d5;
int i75;
int $i76;
int $i77;
int $i78;
int i79;
double $d6;
double $d7;
double $d8;
int $i80;
int i81;
int $i82;
int i83;
int i84;
int i85;
int $i86;
int $i87;
int $i88;
int i89;
double $d9;
double $d10;
double $d11;
long long l90;
int i91;
int $i92;
int i93;
int i94;
double d12;
int i95;
int $i96;
int $i97;
int $i98;
int i99;
int $r69 = -1;
int $r70 = -1;
double $d13;
double $d14;
double $d15;
double $d16;
int $r71 = -1;
int $r72 = -1;
char $b100;
int i101;
int $i102;
int $i103;
int $i104;
int i105;
int $r73 = -1;
int $r74 = -1;
double $d17;
long long $l106;
int $r75 = -1;
int $r76 = -1;
long long $l107;
long long $l108;
char $b109;
int $i110;
int $i111;
long long $l112;
long long $l113;
char $b114;
int $r77 = -1;
int $r78 = -1;
 r0  =  thisref ;
 i0  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getBlockDimx(gc_info, exception);
if(*exception != 0) {
 
return ; }
 i1  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getThreadIdxx(gc_info, exception);
if(*exception != 0) {
 
return ; }
 i2  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getThreadId(gc_info, exception);
if(*exception != 0) {
 
return ; }
 $r6  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 i3  = edu_syr_pcpratts_array_length(gc_info,  $r6 );
 $r7  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r8  = double__array__array_get(gc_info, $r7, 0, exception);
if(*exception != 0) {
 
return ; }
 i4  = edu_syr_pcpratts_array_length(gc_info,  $r8 );
 $i20  =  i3  *  i4  ;
 i5  =  $i20  *  8  ;
 $i21  =  i3  *  i4  ;
 $i22  =  $i21  *  8  ;
 i6  =  i5  +  $i22  ;
 $i23  =  i3  *  4  ;
 i7  =  i6  +  $i23  ;
 $i24  =  i0  *  i4  ;
 $i25  =  $i24  *  8  ;
 i8  =  i7  +  $i25  ;
 $r9  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_9_(gc_info,
 $r9,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "SharedMemorySize: ", exception) , exception);
if(*exception != 0) {
 
return ; }
 $r10  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_5_(gc_info,
 $r10,
  i8 , exception);
if(*exception != 0) {
 
return ; }
 $r11  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_9_(gc_info,
 $r11,
  edu_syr_pcpratts_string_constant(gc_info, (char *) " bytes", exception) , exception);
if(*exception != 0) {
 
return ; }
label67:
 $r12  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_9_(gc_info,
 $r12,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "Start loop...", exception) , exception);
if(*exception != 0) {
 
return ; }
if ( i1  !=  0   ) goto label0;
 i9  =  0 ;
label4:
if ( i9  >=  i3   ) goto label0;
 i10  =  0 ;
label3:
if ( i10  >=  i4   ) goto label2;
 $i26  =  i9  *  i4  ;
 $i27  =  $i26  +  i10  ;
 i11  =  $i27  *  8  ;
 $r13  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r14  = double__array__array_get(gc_info, $r13, i9, exception);
if(*exception != 0) {
 
return ; }
 $d0  = double__array_get(gc_info, $r14, i10, exception);
if(*exception != 0) {
 
return ; }
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedDouble(gc_info,  i11 ,  $d0 , exception);
if(*exception != 0) {
 
return ; }
 $i28  =  i9  *  i4  ;
 $i29  =  $i28  +  i10  ;
 $i30  =  $i29  *  8  ;
 i12  =  i5  +  $i30  ;
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedDouble(gc_info,  i12 ,  0.0 , exception);
if(*exception != 0) {
 
return ; }
 i10  =  i10  +  1  ;
goto label3;
label2:
 $i31  =  i9  *  4  ;
 i32  =  i6  +  $i31  ;
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedInteger(gc_info,  i32 ,  -1 , exception);
if(*exception != 0) {
 
return ; }
 i9  =  i9  +  1  ;
goto label4;
label0:
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_syncthreads(gc_info, exception);
if(*exception != 0) {
 
return ; }
 z0  =  1 ;
 z1  =  0 ;
 i33  =  0 ;
label23:
if ( z0  ==  0   ) goto label5;
 i34  =  0 ;
if ( i1  !=  0   ) goto label6;
 $r15  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_cache(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
if ( $r15  ==  -1   ) goto label7;
if ( z1  ==  0   ) goto label8;
label7:
 $r16  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_cache(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
if ( $r16  !=  -1   ) goto label9;
 $r17  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r17, at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_cache(gc_info, r0,  $r17 , exception);
if(*exception != 0) {
 
return ; }
 z1  =  1 ;
label9:
 r1  =  edu_syr_pcpratts_string_constant(gc_info, (char *) "", exception) ;
 $r18  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r18, edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_initab850b60f96d11de8a390800200c9a660_13_13_(gc_info,
  r1 ,
  -1 , exception));
if(*exception != 0) {
 
return ; }
 r2  =  $r18 ;
label15:
if ( i34  >=  i0   ) goto label10;
 z0  = edu_syr_pcpratts_rootbeer_runtime_HamaPeer_readNext(gc_info,  r2 , exception);
if(*exception != 0) {
 
return ; }
 z1  =  z0 ;
if ( z0  !=  0   ) goto label11;
goto label10;
label11:
 $r19  = edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_getKey13_(gc_info,
 r2, exception);
if(*exception != 0) {
 
return ; }
 r20  = (int)  $r19 ;
 $r21  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r21, at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_initab850b60f96d11de8a390800200c9a660_9_(gc_info,
  r20 , exception));
if(*exception != 0) {
 
return ; }
 r3  =  $r21 ;
 $r22  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_cache(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_add0_11_(gc_info,
 $r22,
  r3 , exception);
if(*exception != 0) {
 
return ; }
 r4  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_toArraya12_(gc_info,
 r3, exception);
if(*exception != 0) {
 
return ; }
 i13  =  0 ;
label14:
if ( i13  >=  i4   ) goto label13;
 $i35  =  i34  *  i4  ;
 $i36  =  $i35  +  i13  ;
 $i37  =  $i36  *  8  ;
 i14  =  i7  +  $i37  ;
 $d1  = double__array_get(gc_info, r4, i13, exception);
if(*exception != 0) {
 
return ; }
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedDouble(gc_info,  i14 ,  $d1 , exception);
if(*exception != 0) {
 
return ; }
 i13  =  i13  +  1  ;
goto label14;
label13:
 i34  =  i34  +  1  ;
goto label15;
label10:
goto label6;
label8:
 i38  =  i33 ;
label22:
if ( i34  >=  i0   ) goto label17;
 $r23  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_cache(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 r24  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_get11_5_(gc_info,
 $r23,
  i38 , exception);
if(*exception != 0) {
 
return ; }
 r25  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_toArraya12_(gc_info,
 r24, exception);
if(*exception != 0) {
 
return ; }
 i39  =  0 ;
label19:
if ( i39  >=  i4   ) goto label18;
 $i40  =  i34  *  i4  ;
 $i41  =  $i40  +  i39  ;
 $i42  =  $i41  *  8  ;
 i43  =  i7  +  $i42  ;
 $d2  = double__array_get(gc_info, r25, i39, exception);
if(*exception != 0) {
 
return ; }
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedDouble(gc_info,  i43 ,  $d2 , exception);
if(*exception != 0) {
 
return ; }
 i39  =  i39  +  1  ;
goto label19;
label18:
 i34  =  i34  +  1  ;
 i38  =  i38  +  1  ;
 $r26  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_cache(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i44  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_getLength5_(gc_info,
 $r26, exception);
if(*exception != 0) {
 
return ; }
if ( i38  !=  $i44   ) goto label20;
 z0  =  0 ;
goto label17;
label20:
goto label22;
label17:
 i33  =  i38 ;
label6:
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_syncthreads(gc_info, exception);
if(*exception != 0) {
 
return ; }
 i45  = at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_getNearestCenter5_5_5_5_(gc_info,
 r0,
  i3 ,
  i4 ,
  i7 , exception);
if(*exception != 0) {
 
return ; }
at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_assignCenters0_5_5_5_5_5_(gc_info,
 r0,
  i45 ,
  i4 ,
  i5 ,
  i6 ,
  i7 , exception);
if(*exception != 0) {
 
return ; }
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_syncthreads(gc_info, exception);
if(*exception != 0) {
 
return ; }
goto label23;
label5:
if ( i1  !=  0   ) goto label24;
 r27  = edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getAllPeerNames(gc_info, exception);
if(*exception != 0) {
 
return ; }
 i46  =  0 ;
label32:
if ( i46  >=  i3   ) goto label24;
 $i47  =  i46  *  4  ;
 i48  =  i6  +  $i47  ;
 i49  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedInteger(gc_info,  i48 , exception);
if(*exception != 0) {
 
return ; }
if ( i49  ==  -1   ) goto label26;
 r28  =  edu_syr_pcpratts_string_constant(gc_info, (char *) "", exception) ;
 $r29  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r29, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
 $r30  = java_lang_StringBuilder_append10_9_(gc_info,
 $r29,
  r28 , exception);
if(*exception != 0) {
 
return ; }
 $r31  = java_lang_Integer_toString9_5_(gc_info,  i46 , exception);
if(*exception != 0) {
 
return ; }
 $r32  = java_lang_StringBuilder_append10_9_(gc_info,
 $r30,
  $r31 , exception);
if(*exception != 0) {
 
return ; }
 r33  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r32, exception);
if(*exception != 0) {
 
return ; }
 $r34  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r34, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
 $r35  = java_lang_StringBuilder_append10_9_(gc_info,
 $r34,
  r33 , exception);
if(*exception != 0) {
 
return ; }
 $r36  = java_lang_StringBuilder_append10_9_(gc_info,
 $r35,
  edu_syr_pcpratts_string_constant(gc_info, (char *) ":", exception) , exception);
if(*exception != 0) {
 
return ; }
 r37  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r36, exception);
if(*exception != 0) {
 
return ; }
 $r38  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r38, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
 $r39  = java_lang_StringBuilder_append10_9_(gc_info,
 $r38,
  r37 , exception);
if(*exception != 0) {
 
return ; }
 $r40  = java_lang_Integer_toString9_5_(gc_info,  i49 , exception);
if(*exception != 0) {
 
return ; }
 $r41  = java_lang_StringBuilder_append10_9_(gc_info,
 $r39,
  $r40 , exception);
if(*exception != 0) {
 
return ; }
 r42  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r41, exception);
if(*exception != 0) {
 
return ; }
 $r43  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r43, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
 $r44  = java_lang_StringBuilder_append10_9_(gc_info,
 $r43,
  r42 , exception);
if(*exception != 0) {
 
return ; }
 $r45  = java_lang_StringBuilder_append10_9_(gc_info,
 $r44,
  edu_syr_pcpratts_string_constant(gc_info, (char *) ":", exception) , exception);
if(*exception != 0) {
 
return ; }
 r46  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r45, exception);
if(*exception != 0) {
 
return ; }
 i50  =  0 ;
label29:
if ( i50  >=  i4   ) goto label27;
 $i51  =  i46  *  i4  ;
 $i52  =  $i51  +  i50  ;
 $i53  =  $i52  *  8  ;
 i54  =  i5  +  $i53  ;
 $r47  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r47, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
 $r48  = java_lang_StringBuilder_append10_9_(gc_info,
 $r47,
  r46 , exception);
if(*exception != 0) {
 
return ; }
 $d3  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedDouble(gc_info,  i54 , exception);
if(*exception != 0) {
 
return ; }
 $r49  = java_lang_Double_toString9_8_(gc_info,  $d3 , exception);
if(*exception != 0) {
 
return ; }
 $r50  = java_lang_StringBuilder_append10_9_(gc_info,
 $r48,
  $r49 , exception);
if(*exception != 0) {
 
return ; }
 r46  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r50, exception);
if(*exception != 0) {
 
return ; }
 $i55  =  i4  -  1  ;
if ( i50  >=  $i55   ) goto label28;
 $r51  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r51, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
 $r52  = java_lang_StringBuilder_append10_9_(gc_info,
 $r51,
  r46 , exception);
if(*exception != 0) {
 
return ; }
 $r53  = java_lang_StringBuilder_append10_9_(gc_info,
 $r52,
  edu_syr_pcpratts_string_constant(gc_info, (char *) ", ", exception) , exception);
if(*exception != 0) {
 
return ; }
 r46  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r53, exception);
if(*exception != 0) {
 
return ; }
label28:
 i50  =  i50  +  1  ;
goto label29;
label27:
 $r54  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_9_(gc_info,
 $r54,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "send message: \'", exception) , exception);
if(*exception != 0) {
 
return ; }
 $r55  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_9_(gc_info,
 $r55,
  r46 , exception);
if(*exception != 0) {
 
return ; }
 $r56  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_9_(gc_info,
 $r56,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "\'", exception) , exception);
if(*exception != 0) {
 
return ; }
 r57  =  r27 ;
 i56  = edu_syr_pcpratts_array_length(gc_info,  r57 );
 i15  =  0 ;
label31:
if ( i15  >=  i56   ) goto label26;
 r5  = java_lang_String__array_get(gc_info, r57, i15, exception);
if(*exception != 0) {
 
return ; }
edu_syr_pcpratts_rootbeer_runtime_HamaPeer_send(gc_info,  r5 ,  r46 , exception);
if(*exception != 0) {
 
return ; }
 i15  =  i15  +  1  ;
goto label31;
label26:
 i46  =  i46  +  1  ;
goto label32;
label24:
if ( i2  !=  0   ) goto label33;
edu_syr_pcpratts_rootbeer_runtime_HamaPeer_sync(gc_info, exception);
if(*exception != 0) {
 
return ; }
label33:
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_syncthreads(gc_info, exception);
if(*exception != 0) {
 
return ; }
if ( i2  !=  0   ) goto label34;
 i57  =  0 ;
label38:
if ( i57  >=  i3   ) goto label35;
 i58  =  0 ;
label37:
if ( i58  >=  i4   ) goto label36;
 $i59  =  i57  *  i4  ;
 $i60  =  $i59  +  i58  ;
 $i61  =  $i60  *  8  ;
 i62  =  i5  +  $i61  ;
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedDouble(gc_info,  i62 ,  0.0 , exception);
if(*exception != 0) {
 
return ; }
 i58  =  i58  +  1  ;
goto label37;
label36:
 $i63  =  i57  *  4  ;
 i64  =  i6  +  $i63  ;
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedInteger(gc_info,  i64 ,  0 , exception);
if(*exception != 0) {
 
return ; }
 i57  =  i57  +  1  ;
goto label38;
label35:
 i65  = edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getNumCurrentMessages(gc_info, exception);
if(*exception != 0) {
 
return ; }
 i66  =  0 ;
label48:
if ( i66  >=  i65   ) goto label39;
 r58  = edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getCurrentStringMessage(gc_info, exception);
if(*exception != 0) {
 
return ; }
 $r59  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_9_(gc_info,
 $r59,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "got message: \'", exception) , exception);
if(*exception != 0) {
 
return ; }
 $r60  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_9_(gc_info,
 $r60,
  r58 , exception);
if(*exception != 0) {
 
return ; }
 $r61  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_9_(gc_info,
 $r61,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "\'", exception) , exception);
if(*exception != 0) {
 
return ; }
 r62  = java_lang_String_split(gc_info,
 r58,
  edu_syr_pcpratts_string_constant(gc_info, (char *) ":", exception) ,
  3 , exception);
if(*exception != 0) {
 
return ; }
 $r63  = java_lang_String__array_get(gc_info, r62, 0, exception);
if(*exception != 0) {
 
return ; }
 i67  = java_lang_Integer_parseInt(gc_info,  $r63 , exception);
if(*exception != 0) {
 
return ; }
 $r64  = java_lang_String__array_get(gc_info, r62, 1, exception);
if(*exception != 0) {
 
return ; }
 i68  = java_lang_Integer_parseInt(gc_info,  $r64 , exception);
if(*exception != 0) {
 
return ; }
 $r65  = java_lang_String__array_get(gc_info, r62, 2, exception);
if(*exception != 0) {
 
return ; }
 r66  = java_lang_String_split(gc_info,
 $r65,
  edu_syr_pcpratts_string_constant(gc_info, (char *) ",", exception) , exception);
if(*exception != 0) {
 
return ; }
 i69  = edu_syr_pcpratts_array_length(gc_info,  r66 );
 r67  = double__array_new(gc_info,  i69 , exception);
 i16  =  0 ;
label41:
if ( i16  >=  i69   ) goto label40;
 $r68  = java_lang_String__array_get(gc_info, r66, i16, exception);
if(*exception != 0) {
 
return ; }
 $d4  = java_lang_Double_parseDouble(gc_info,  $r68 , exception);
if(*exception != 0) {
 
return ; }
double__array_set(gc_info, r67, i16,  $d4 , exception);
if(*exception != 0) {
 
return ; }
 i16  =  i16  +  1  ;
goto label41;
label40:
 $i70  =  i67  *  4  ;
 i71  =  i6  +  $i70  ;
 i17  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedInteger(gc_info,  i71 , exception);
if(*exception != 0) {
 
return ; }
if ( i17  !=  0   ) goto label42;
 i18  =  0 ;
label44:
if ( i18  >=  i4   ) goto label43;
 $i72  =  i66  *  i4  ;
 $i73  =  $i72  +  i18  ;
 $i74  =  $i73  *  8  ;
 i19  =  i5  +  $i74  ;
 $d5  = double__array_get(gc_info, r67, i18, exception);
if(*exception != 0) {
 
return ; }
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedDouble(gc_info,  i19 ,  $d5 , exception);
if(*exception != 0) {
 
return ; }
 i18  =  i18  +  1  ;
goto label44;
label43:
goto label45;
label42:
 i75  =  0 ;
label47:
if ( i75  >=  i4   ) goto label45;
 $i76  =  i66  *  i4  ;
 $i77  =  $i76  +  i75  ;
 $i78  =  $i77  *  8  ;
 i79  =  i5  +  $i78  ;
 $d6  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedDouble(gc_info,  i79 , exception);
if(*exception != 0) {
 
return ; }
 $d7  = double__array_get(gc_info, r67, i75, exception);
if(*exception != 0) {
 
return ; }
 $d8  =  $d6  +  $d7  ;
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedDouble(gc_info,  i79 ,  $d8 , exception);
if(*exception != 0) {
 
return ; }
 i75  =  i75  +  1  ;
goto label47;
label45:
 $i80  =  i17  +  i68  ;
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedInteger(gc_info,  i71 ,  $i80 , exception);
if(*exception != 0) {
 
return ; }
 i66  =  i66  +  1  ;
goto label48;
label39:
 i81  =  0 ;
label53:
if ( i81  >=  i3   ) goto label49;
 $i82  =  i81  *  4  ;
 i83  =  i6  +  $i82  ;
 i84  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedInteger(gc_info,  i83 , exception);
if(*exception != 0) {
 
return ; }
if ( i84  ==  0   ) goto label50;
 i85  =  0 ;
label52:
if ( i85  >=  i4   ) goto label50;
 $i86  =  i81  *  i4  ;
 $i87  =  $i86  +  i85  ;
 $i88  =  $i87  *  8  ;
 i89  =  i5  +  $i88  ;
 $d9  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedDouble(gc_info,  i89 , exception);
if(*exception != 0) {
 
return ; }
 $d10  = (double)  i84 ;
 $d11  =  $d9  /  $d10  ;
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_setSharedDouble(gc_info,  i89 ,  $d11 , exception);
if(*exception != 0) {
 
return ; }
 i85  =  i85  +  1  ;
goto label52;
label50:
 i81  =  i81  +  1  ;
goto label53;
label49:
 l90  =  0L ;
 i91  =  0 ;
label61:
if ( i91  >=  i3   ) goto label54;
 $i92  =  i91  *  4  ;
 i93  =  i6  +  $i92  ;
 i94  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedInteger(gc_info,  i93 , exception);
if(*exception != 0) {
 
return ; }
if ( i94  ==  0   ) goto label55;
 d12  =  0.0 ;
 i95  =  0 ;
label57:
if ( i95  >=  i4   ) goto label56;
 $i96  =  i91  *  i4  ;
 $i97  =  $i96  +  i95  ;
 $i98  =  $i97  *  8  ;
 i99  =  i5  +  $i98  ;
 $r69  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r70  = double__array__array_get(gc_info, $r69, i91, exception);
if(*exception != 0) {
 
return ; }
 $d13  = double__array_get(gc_info, $r70, i95, exception);
if(*exception != 0) {
 
return ; }
 $d14  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedDouble(gc_info,  i99 , exception);
if(*exception != 0) {
 
return ; }
 $d15  =  $d13  -  $d14  ;
 $d16  = java_lang_Math_abs8_8_(gc_info,  $d15 , exception);
if(*exception != 0) {
 
return ; }
 d12  =  d12  +  $d16  ;
 i95  =  i95  +  1  ;
goto label57;
label56:
 $r71  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_9_(gc_info,
 $r71,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "calculateError: ", exception) , exception);
if(*exception != 0) {
 
return ; }
 $r72  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_8_(gc_info,
 $r72,
  d12 , exception);
if(*exception != 0) {
 
return ; }
 $b100  = edu_syr_pcpratts_cmpl((double) d12 , (double) 0.0 );
if ( $b100  <=  0   ) goto label55;
 i101  =  0 ;
label60:
if ( i101  >=  i4   ) goto label59;
 $i102  =  i91  *  i4  ;
 $i103  =  $i102  +  i101  ;
 $i104  =  $i103  *  8  ;
 i105  =  i5  +  $i104  ;
 $r73  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r74  = double__array__array_get(gc_info, $r73, i91, exception);
if(*exception != 0) {
 
return ; }
 $d17  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getSharedDouble(gc_info,  i105 , exception);
if(*exception != 0) {
 
return ; }
double__array_set(gc_info, $r74, i101,  $d17 , exception);
if(*exception != 0) {
 
return ; }
 i101  =  i101  +  1  ;
goto label60;
label59:
 l90  =  l90  +  1L  ;
label55:
 i91  =  i91  +  1  ;
goto label61;
label54:
instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_converged(gc_info, r0,  l90 , exception);
if(*exception != 0) {
 
return ; }
 $l106  = edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getSuperstepCount(gc_info, exception);
if(*exception != 0) {
 
return ; }
instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_superstepCount(gc_info, r0,  $l106 , exception);
if(*exception != 0) {
 
return ; }
label34:
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_syncthreads(gc_info, exception);
if(*exception != 0) {
 
return ; }
 $r75  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_9_(gc_info,
 $r75,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "m_converged: ", exception) , exception);
if(*exception != 0) {
 
return ; }
 $r76  = static_getter_java_lang_System_out(gc_info, exception);
 $l107  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_converged(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
java_io_PrintStream_println0_6_(gc_info,
 $r76,
  $l107 , exception);
if(*exception != 0) {
 
return ; }
 $l108  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_converged(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $b109  = edu_syr_pcpratts_cmp( $l108 ,  0L );
if ( $b109  !=  0   ) goto label62;
goto label63;
label62:
 $i110  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_maxIterations(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
if ( $i110  <=  0   ) goto label64;
 $i111  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_maxIterations(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $l112  = (long long)  $i111 ;
 $l113  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_superstepCount(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $b114  = edu_syr_pcpratts_cmp( $l112 ,  $l113 );
if ( $b114  >=  0   ) goto label64;
goto label63;
label64:
goto label67;
label63:
 $r77  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_9_(gc_info,
 $r77,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "Finished! Writing the assignments...", exception) , exception);
if(*exception != 0) {
 
return ; }
 $r78  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_9_(gc_info,
 $r78,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "Done.", exception) , exception);
if(*exception != 0) {
 
return ; }
return;
}
__device__ int invoke_java_lang_StringBuilder_toString9_( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
GC_OBJ_TYPE_TYPE derived_type;
if(thisref == -1){
  *exception = -2;
return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
derived_type = edu_syr_pcpratts_gc_get_type(thisref_deref);
if(0){
}
else if(derived_type == 1){
return java_lang_Object_toString9_(gc_info, thisref, exception);
}
else if(derived_type == 16901){
return java_lang_StringBuilder_toString9_(gc_info, thisref, exception);
}
return -1;
}
__device__ int java_lang_IllegalArgumentException_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 32));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 21377);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 32);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_java_lang_Throwable_cause(gc_info, thisref, -1, exception);
instance_setter_java_lang_Throwable_detailMessage(gc_info, thisref, -1, exception);
 r0  =  thisref ;
 r1  =  parameter0 ;
java_lang_RuntimeException_initab850b60f96d11de8a390800200c9a66_body0_9_(gc_info,
 thisref,
  r1 , exception);
return r0;
  return 0;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_get11_5_( char * gc_info, int thisref, int parameter0, int * exception){
int r0 = -1;
int i0;
int $r1 = -1;
int $r2 = -1;
 r0  =  thisref ;
 i0  =  parameter0 ;
 $r1  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 $r2  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_get(gc_info, $r1, i0, exception);
if(*exception != 0) {
 
return 0; }
return  $r2 ;
  return 0;
}
__device__ int java_lang_String_hashCode5_( char * gc_info, int thisref, int * exception){
int r0 = -1;
int i0;
int i1;
int i2;
int r1 = -1;
int i3;
int $i4;
int $i5;
char $c6;
 r0  =  thisref ;
 i0  = instance_getter_java_lang_String_hash(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 i1  = instance_getter_java_lang_String_count(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
if ( i0  !=  0   ) goto label0;
if ( i1  <=  0   ) goto label0;
 i2  = instance_getter_java_lang_String_offset(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 r1  = instance_getter_java_lang_String_value(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 i3  =  0 ;
label3:
if ( i3  >=  i1   ) goto label2;
 $i5  =  31  *  i0  ;
 $i4  =  i2 ;
 i2  =  i2  +  1  ;
 $c6  = char__array_get(gc_info, r1, $i4, exception);
if(*exception != 0) {
 
return 0; }
 i0  =  $i5  +  $c6  ;
 i3  =  i3  +  1  ;
goto label3;
label2:
instance_setter_java_lang_String_hash(gc_info, r0,  i0 , exception);
if(*exception != 0) {
 
return 0; }
label0:
return  i0 ;
  return 0;
}
__device__ int edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception){
int i0;
int i1;
int i2;
int $r0 = -1;
int r1 = -1;
 i0  =  parameter0 ;
 i1  =  parameter1 ;
 i2  =  parameter2 ;
 $r0  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r0, edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return 0; }
 r1  =  $r0 ;
instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayLength(gc_info, r1,  i2 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayIndex(gc_info, r1,  i0 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_array(gc_info, r1,  i1 , exception);
if(*exception != 0) {
 
return 0; }
return  r1 ;
  return 0;
}
__device__ int java_util_Arrays_copyOfRangea14_a14_5_5_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception){
int r0 = -1;
int i0;
int i1;
int i2;
int $r1 = -1;
int r2 = -1;
int $r3 = -1;
int $r4 = -1;
int $r5 = -1;
int $r6 = -1;
int $r7 = -1;
int $i3;
int $i4;
int $i5;
 r0  =  parameter0 ;
 i0  =  parameter1 ;
 i1  =  parameter2 ;
 i2  =  i1  -  i0  ;
if ( i2  >=  0   ) goto label0;
 $r3  =  -1 ;
 $r1  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r1, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return 0; }
 $r4  = java_lang_StringBuilder_append10_5_(gc_info,
 $r1,
  i0 , exception);
if(*exception != 0) {
 
return 0; }
 $r5  = java_lang_StringBuilder_append10_9_(gc_info,
 $r4,
  edu_syr_pcpratts_string_constant(gc_info, (char *) " > ", exception) , exception);
if(*exception != 0) {
 
return 0; }
 $r6  = java_lang_StringBuilder_append10_5_(gc_info,
 $r5,
  i1 , exception);
if(*exception != 0) {
 
return 0; }
 $r7  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r6, exception);
if(*exception != 0) {
 
return 0; }
edu_syr_pcpratts_gc_assign (gc_info, 
&$r3, java_lang_IllegalArgumentException_initab850b60f96d11de8a390800200c9a660_9_(gc_info,
  $r7 , exception));
if(*exception != 0) {
 
return 0; }
 *exception =  $r3 ;
return 0;
label0:
 r2  = char__array_new(gc_info,  i2 , exception);
 $i3  = edu_syr_pcpratts_array_length(gc_info,  r0 );
 $i4  =  $i3  -  i0  ;
 $i5  = java_lang_Math_min5_5_5_(gc_info,  $i4 ,  i2 , exception);
if(*exception != 0) {
 
return 0; }
java_lang_System_arraycopy(gc_info,  r0 ,  i0 ,  r2 ,  0 ,  $i5 , exception);
if(*exception != 0) {
 
return 0; }
return  r2 ;
  return 0;
}
__device__ double java_lang_Math_sqrt8_8_( char * gc_info, double parameter0, int * exception){
double d0;
double $d1;
 d0  =  parameter0 ;
 $d1  = java_lang_StrictMath_sqrt(gc_info,  d0 , exception);
if(*exception != 0) {
 
return 0; }
return  $d1 ;
  return 0;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
int r2 = -1;
int i0;
int $i1;
int $r3 = -1;
int $i2;
int $r4 = -1;
int $r5 = -1;
double $d0;
int $i3;
int $r6 = -1;
int $r7 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 48));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 1);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 2903);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 48);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector(gc_info, thisref, -1, exception);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index(gc_info, thisref, 0, exception);
 r0  =  thisref ;
 r1  =  parameter0 ;
 r2  = java_lang_String_split(gc_info,
 r1,
  edu_syr_pcpratts_string_constant(gc_info, (char *) ",", exception) , exception);
if(*exception != 0) {
 
return 0; }
if ( r2  ==  -1   ) goto label0;
 $i1  = edu_syr_pcpratts_array_length(gc_info,  r2 );
 $r3  = double__array_new(gc_info,  $i1 , exception);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector(gc_info, r0,  $r3 , exception);
if(*exception != 0) {
 
return 0; }
 i0  =  0 ;
label2:
 $i2  = edu_syr_pcpratts_array_length(gc_info,  r2 );
if ( i0  >=  $i2   ) goto label1;
 $r4  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 $r5  = java_lang_String__array_get(gc_info, r2, i0, exception);
if(*exception != 0) {
 
return 0; }
 $d0  = java_lang_Double_parseDouble(gc_info,  $r5 , exception);
if(*exception != 0) {
 
return 0; }
double__array_set(gc_info, $r4, i0,  $d0 , exception);
if(*exception != 0) {
 
return 0; }
 i0  =  i0  +  1  ;
goto label2;
label1:
 $i3  = edu_syr_pcpratts_array_length(gc_info,  r2 );
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index(gc_info, r0,  $i3 , exception);
if(*exception != 0) {
 
return 0; }
goto label3;
label0:
 $r6  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_9_(gc_info,
 $r6,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "DenseDoubleVector no values found!", exception) , exception);
if(*exception != 0) {
 
return 0; }
 $r7  = double__array_new(gc_info,  128 , exception);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector(gc_info, r0,  $r7 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index(gc_info, r0,  0 , exception);
if(*exception != 0) {
 
return 0; }
label3:
return r0;
  return 0;
}
__device__ int java_lang_Object_toString9_( char * gc_info, int thisref, int * exception){
int r0 = -1;
int $r1 = -1;
int $r2 = -1;
int $r3 = -1;
int $r4 = -1;
int $r5 = -1;
int $i0;
int $r6 = -1;
int $r7 = -1;
int $r8 = -1;
 r0  =  thisref ;
 $r1  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r1, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return 0; }
 $r2  = java_lang_Object_getClass(gc_info,
 r0, exception);
if(*exception != 0) {
 
return 0; }
 $r3  = java_lang_Class_getName(gc_info,
 $r2, exception);
if(*exception != 0) {
 
return 0; }
 $r4  = java_lang_StringBuilder_append10_9_(gc_info,
 $r1,
  $r3 , exception);
if(*exception != 0) {
 
return 0; }
 $r5  = java_lang_StringBuilder_append10_9_(gc_info,
 $r4,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "@", exception) , exception);
if(*exception != 0) {
 
return 0; }
 $i0  = invoke_java_lang_Object_hashCode(gc_info,
 r0, exception);
if(*exception != 0) {
 
return 0; }
 $r6  = java_lang_Integer_toHexString9_5_(gc_info,  $i0 , exception);
if(*exception != 0) {
 
return 0; }
 $r7  = java_lang_StringBuilder_append10_9_(gc_info,
 $r5,
  $r6 , exception);
if(*exception != 0) {
 
return 0; }
 $r8  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r7, exception);
if(*exception != 0) {
 
return 0; }
return  $r8 ;
  return 0;
}
__device__ int edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception){
int r0 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 48));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 3608);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 48);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_array(gc_info, thisref, 0, exception);
instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayIndex(gc_info, thisref, 0, exception);
instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayLength(gc_info, thisref, 0, exception);
 r0  =  thisref ;
return r0;
  return 0;
}
__device__ int java_lang_Integer_stringSize5_5_( char * gc_info, int parameter0, int * exception){
int i0;
int i1;
int $r0 = -1;
int $i2;
int $i3;
 i0  =  parameter0 ;
 i1  =  0 ;
label1:
 $r0  = static_getter_java_lang_Integer_sizeTable(gc_info, exception);
 $i2  = int__array_get(gc_info, $r0, i1, exception);
if(*exception != 0) {
 
return 0; }
if ( i0  >  $i2   ) goto label0;
 $i3  =  i1  +  1  ;
return  $i3 ;
label0:
 i1  =  i1  +  1  ;
goto label1;
  return 0;
}
__device__ double double__array_get( char * gc_info, int thisref, int parameter0, int * exception){
int offset;
int length;
 char * thisref_deref;
offset = 32+(parameter0*8);
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return 0;
}
return *(( double *) &thisref_deref[offset]);
}
__device__ void double__array_set( char * gc_info, int thisref, int parameter0, double parameter1, int * exception){
int length;
 char * thisref_deref;
  if(thisref == -1){
    *exception = 21352;
    return;
  }
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return;
}
*(( double *) &thisref_deref[32+(parameter0*8)]) = parameter1;
}
__device__ int double__array_new( char * gc_info, int size, int * exception){
int i;
int total_size;
int mod;
int thisref;
 char * thisref_deref;
total_size = (size * 8)+ 32;
mod = total_size % 8;
if(mod != 0)
  total_size += (8 - mod);
thisref = edu_syr_pcpratts_gc_malloc(gc_info, total_size);
if(thisref == -1){
  *exception = 21352;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 2909);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, total_size);
edu_syr_pcpratts_setint(thisref_deref, 12, size);
for(i = 0; i < size; ++i){
  double__array_set(gc_info, thisref, i, 0, exception);
}
return thisref;
}
__device__ int java_lang_Integer_toHexString9_5_( char * gc_info, int parameter0, int * exception){
int i0;
int $r0 = -1;
 i0  =  parameter0 ;
 $r0  = java_lang_Integer_toUnsignedString9_5_5_(gc_info,  i0 ,  4 , exception);
if(*exception != 0) {
 
return 0; }
return  $r0 ;
  return 0;
}
__device__ int java_lang_Math_min5_5_5_( char * gc_info, int parameter0, int parameter1, int * exception){
int i0;
int i1;
int $i2;
 i0  =  parameter0 ;
 i1  =  parameter1 ;
if ( i0  >  i1   ) goto label0;
 $i2  =  i0 ;
goto label1;
label0:
 $i2  =  i1 ;
label1:
return  $i2 ;
  return 0;
}
__device__ int java_lang_StringIndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a660_5_( char * gc_info, int parameter0, int * exception){
int r0 = -1;
int i0;
int $r1 = -1;
int $r2 = -1;
int $r3 = -1;
int $r4 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 32));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 22978);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 32);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_java_lang_Throwable_cause(gc_info, thisref, -1, exception);
instance_setter_java_lang_Throwable_detailMessage(gc_info, thisref, -1, exception);
 r0  =  thisref ;
 i0  =  parameter0 ;
 $r1  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r1, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return 0; }
 $r2  = java_lang_StringBuilder_append10_9_(gc_info,
 $r1,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "String index out of range: ", exception) , exception);
if(*exception != 0) {
 
return 0; }
 $r3  = java_lang_StringBuilder_append10_5_(gc_info,
 $r2,
  i0 , exception);
if(*exception != 0) {
 
return 0; }
 $r4  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r3, exception);
if(*exception != 0) {
 
return 0; }
java_lang_IndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a66_body0_9_(gc_info,
 thisref,
  $r4 , exception);
return r0;
  return 0;
}
__device__ int instance_getter_java_lang_String_hash( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[44]);
}
__device__ void instance_setter_java_lang_String_hash( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[44]) = parameter0;
}
__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayLength( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[40]) = parameter0;
}
__device__ int instance_getter_java_lang_AbstractStringBuilder_count( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[40]);
}
__device__ void instance_setter_java_lang_AbstractStringBuilder_count( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[40]) = parameter0;
}
__device__ long long instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_converged( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( long long *) &thisref_deref[40]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_converged( char * gc_info, int thisref, long long parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( long long *) &thisref_deref[40]) = parameter0;
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_maxIterations( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[56]);
}
__device__ int static_getter_java_lang_System_out( char * gc_info, int * exception){
 char * thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, 0);
return *(( int *) &thisref_deref[12]);
}
__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayIndex( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[36]) = parameter0;
}
__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[36]);
}
__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[36]) = parameter0;
}
__device__ int instance_getter_java_lang_Class_name( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[40]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[40]) = parameter0;
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ int static_getter_java_lang_Integer_sizeTable( char * gc_info, int * exception){
 char * thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, 0);
return *(( int *) &thisref_deref[8]);
}
__device__ int static_getter_java_lang_Integer_DigitOnes( char * gc_info, int * exception){
 char * thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, 0);
return *(( int *) &thisref_deref[16]);
}
__device__ int static_getter_java_lang_Integer_DigitTens( char * gc_info, int * exception){
 char * thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, 0);
return *(( int *) &thisref_deref[20]);
}
__device__ void instance_setter_java_lang_Throwable_cause( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_array( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ int instance_getter_java_lang_String_count( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[40]);
}
__device__ void instance_setter_java_lang_String_count( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[40]) = parameter0;
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_centers( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[36]);
}
__device__ long long instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_superstepCount( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( long long *) &thisref_deref[48]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_superstepCount( char * gc_info, int thisref, long long parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( long long *) &thisref_deref[48]) = parameter0;
}
__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ int instance_getter_java_lang_AbstractStringBuilder_value( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ void instance_setter_java_lang_AbstractStringBuilder_value( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[40]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[40]) = parameter0;
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_cache( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_m_cache( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ int instance_getter_java_lang_String_value( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ void instance_setter_java_lang_String_value( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ int instance_getter_java_lang_String_offset( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[48]);
}
__device__ void instance_setter_java_lang_String_offset( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[48]) = parameter0;
}
__device__ void instance_setter_java_lang_Throwable_detailMessage( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[36]) = parameter0;
}
__device__ int static_getter_java_lang_Integer_digits( char * gc_info, int * exception){
 char * thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, 0);
return *(( int *) &thisref_deref[0]);
}
__device__ void at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_add0_11_( char * gc_info, int thisref, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
int r2 = -1;
int i0;
int $r3 = -1;
int $i1;
int $i2;
int $i3;
int $i4;
int $r5 = -1;
int $i5;
int $i6;
int $i7;
int $i8;
int $r6 = -1;
int $r7 = -1;
 r0  =  thisref ;
 r1  =  parameter0 ;
 $r3  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i1  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_set(gc_info, $r3, $i1,  r1 , exception);
if(*exception != 0) {
 
return ; }
 $i2  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i3  =  $i2  +  1  ;
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0,  $i3 , exception);
if(*exception != 0) {
 
return ; }
 $i4  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r5  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i5  = edu_syr_pcpratts_array_length(gc_info,  $r5 );
if ( $i4  !=  $i5   ) goto label0;
 $i6  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i7  =  $i6  *  2  ;
 r2  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_new(gc_info,  $i7 , exception);
 i0  =  0 ;
label2:
 $i8  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
if ( i0  >=  $i8   ) goto label1;
 $r6  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r7  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_get(gc_info, $r6, i0, exception);
if(*exception != 0) {
 
return ; }
at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_set(gc_info, r2, i0,  $r7 , exception);
if(*exception != 0) {
 
return ; }
 i0  =  i0  +  1  ;
goto label2;
label1:
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, r0,  r2 , exception);
if(*exception != 0) {
 
return ; }
label0:
return;
}
__device__ void java_lang_Integer_getChars0_5_5_a14_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception){
int i0;
int i1;
int r0 = -1;
int i2;
char c3;
int i4;
int i5;
int $i6;
int $i7;
int $i8;
int $i9;
int $i10;
int i11;
int $r1 = -1;
char $c12;
int $r2 = -1;
char $c13;
int $i14;
int i15;
int $i16;
int $i17;
int $i18;
int i19;
int $r3 = -1;
char $c20;
int i21;
 i0  =  parameter0 ;
 i1  =  parameter1 ;
 r0  =  parameter2 ;
 i2  =  i1 ;
 c3  =  0 ;
if ( i0  >=  0   ) goto label0;
 c3  =  45 ;
 i0  = - i0 ;
label0:
if ( i0  <  65536   ) goto label1;
 i4  =  i0  /  100  ;
 $i6  =  i4  <<  6  ;
 $i7  =  i4  <<  5  ;
 $i8  =  $i6  +  $i7  ;
 $i9  =  i4  <<  2  ;
 $i10  =  $i8  +  $i9  ;
 i5  =  i0  -  $i10  ;
 i0  =  i4 ;
 i11  =  i2  +  -1  ;
 $r1  = static_getter_java_lang_Integer_DigitOnes(gc_info, exception);
 $c12  = char__array_get(gc_info, $r1, i5, exception);
if(*exception != 0) {
 
return ; }
char__array_set(gc_info, r0, i11,  $c12 , exception);
if(*exception != 0) {
 
return ; }
 i2  =  i11  +  -1  ;
 $r2  = static_getter_java_lang_Integer_DigitTens(gc_info, exception);
 $c13  = char__array_get(gc_info, $r2, i5, exception);
if(*exception != 0) {
 
return ; }
char__array_set(gc_info, r0, i2,  $c13 , exception);
if(*exception != 0) {
 
return ; }
goto label0;
label1:
 $i14  =  i0  *  52429  ;
 i15  = ( $i14  >>  19  ) & 0x7fffffff;
 $i16  =  i15  <<  3  ;
 $i17  =  i15  <<  1  ;
 $i18  =  $i16  +  $i17  ;
 i19  =  i0  -  $i18  ;
 i2  =  i2  +  -1  ;
 $r3  = static_getter_java_lang_Integer_digits(gc_info, exception);
 $c20  = char__array_get(gc_info, $r3, i19, exception);
if(*exception != 0) {
 
return ; }
char__array_set(gc_info, r0, i2,  $c20 , exception);
if(*exception != 0) {
 
return ; }
 i0  =  i15 ;
if ( i0  !=  0   ) goto label1;
goto label4;
label4:
if ( c3  ==  0   ) goto label5;
 i21  =  i2  +  -1  ;
char__array_set(gc_info, r0, i21,  c3 , exception);
if(*exception != 0) {
 
return ; }
label5:
return;
}
__device__ void java_lang_Throwable_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
 r0  =  thisref ;
 r1  =  parameter0 ;
instance_setter_java_lang_Throwable_cause(gc_info, r0,  r0 , exception);
if(*exception != 0) {
 
return ; }
java_lang_Throwable_fillInStackTrace(gc_info,
 r0, exception);
if(*exception != 0) {
 
return ; }
instance_setter_java_lang_Throwable_detailMessage(gc_info, r0,  r1 , exception);
if(*exception != 0) {
 
return ; }
return;
}
__device__ int
edu_syr_pcpratts_classConstant(int type_num){
  int * temp = (int *) m_Local[2];   
  return temp[type_num];
}
__device__  char *
edu_syr_pcpratts_gc_deref(char * gc_info, int handle){
  char * data_arr = (char * ) m_Local[0];
  long long lhandle = handle;
  lhandle = lhandle << 4;
  return &data_arr[lhandle];
}
__device__ int
edu_syr_pcpratts_gc_malloc(char * gc_info, int size){
  unsigned long long space_size = m_Local[1];
  unsigned long long ret = edu_syr_pcpratts_gc_malloc_no_fail(gc_info, size);
  unsigned long long end = ret + size + 8L;
  if(end >= space_size){
    return -1;
  }
  return (int) (ret >> 4);
}
__device__ unsigned long long
edu_syr_pcpratts_gc_malloc_no_fail(char * gc_info, int size){
  unsigned long long * addr = (unsigned long long *) (gc_info + TO_SPACE_FREE_POINTER_OFFSET);
  if(size % 16 != 0){
    size += (16 - (size %16));
  }
  unsigned long long ret;
  ret = atomicAdd(addr, size);
  return ret;
}
__device__  void
edu_syr_pcpratts_gc_init(char * to_space, size_t space_size, int * java_lang_class_refs){
  if(threadIdx.x == 0){
    m_Local[0] = (size_t) to_space;
    m_Local[1] = (size_t) space_size;
    m_Local[2] = (size_t) java_lang_class_refs;
  }
}
__global__ void entry(char * gc_info, char * to_space, int * handles, 
  long long * to_space_free_ptr, long long * space_size, int * exceptions,
  int * java_lang_class_refs, HostDeviceInterface * h_d_interface,
  int num_blocks) {
  
  host_device_interface = h_d_interface;
  
  
  edu_syr_pcpratts_gc_init(to_space, *space_size, java_lang_class_refs);
  __syncthreads();
  int loop_control = blockIdx.x * blockDim.x + threadIdx.x;
  if(loop_control >= num_blocks){
  
    return;
  }
 else {
    int handle = handles[loop_control];
    int exception = 0;   
    at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernel_gpuMethod0_(gc_info, handle, &exception);
    exceptions[loop_control] = exception;
  
    unsigned long long * addr = ( unsigned long long * ) (gc_info + TO_SPACE_FREE_POINTER_OFFSET);
    *to_space_free_ptr = *addr;
  }
}

