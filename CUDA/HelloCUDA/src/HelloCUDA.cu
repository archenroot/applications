#include "hip/hip_runtime.h"
/**
 * Licensed to the Apache Software Foundation (ASF) under one or more
 * contributor license agreements.  See the NOTICE file distributed with
 * this work for additional information regarding copyright ownership.
 * The ASF licenses this file to You under the Apache License, Version 2.0
 * (the "License"); you may not use this file except in compliance with
 * the License.  You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "util/cuPrintf.cu"
#include <stdio.h>

__global__ void device_greetings(void)
{
  cuPrintf("Hello, world from the device!\n");
}

int main(void)
{
  // greet from the host
  printf("Hello, world from the host!\n");

  // initialize cuPrintf
  cudaPrintfInit();

  // launch a kernel with a single thread to greet from the device
  device_greetings<<<1,1>>>();

  // display the device's greeting
  cudaPrintfDisplay();

  // clean up after cuPrintf
  cudaPrintfEnd();

  return 0;
}
