#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#define N 4096

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

__global__ void increment_kernel(unsigned long long *cnt)
{
  atomicAdd(cnt, 1);
}

int main(int argc, char **argv)
{
  unsigned long long *cnt;

  checkCuda( hipHostMalloc((void**)&cnt, sizeof(unsigned long long)) );

  *cnt = 0;

  for (int i = 0; i < N; i++) {
    increment_kernel<<<4, 4>>>(cnt);
  }

  hipDeviceSynchronize();

  fprintf(stderr, "CNT %lu\n", *cnt);

  assert(*cnt == 16*N);
  fprintf(stderr, "CNT %lu\n", *cnt);

  return 0;
}
