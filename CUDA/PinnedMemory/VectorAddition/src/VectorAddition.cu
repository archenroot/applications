#include "hip/hip_runtime.h"
/**
 * Licensed to the Apache Software Foundation (ASF) under one or more
 * contributor license agreements.  See the NOTICE file distributed with
 * this work for additional information regarding copyright ownership.
 * The ASF licenses this file to You under the Apache License, Version 2.0
 * (the "License"); you may not use this file except in compliance with
 * the License.  You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>

/**
 * Convenience function for checking CUDA runtime API results
 * can be wrapped around any runtime API call. No-op in release builds.
 */
inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}

__global__ void addVector(int *a, int *b, int *c, unsigned int n) {
	int thread_idxx = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_idxx < n) {
		c[thread_idxx] = a[thread_idxx] + b[thread_idxx];
	}
}

void printVector(int *v, unsigned int n) {
	for (int i = 0; i < n; i++) {
		if (i + 1 < n) {
			printf("%d, ", v[i]);
		} else {
			printf("%d", v[i]);
		}
	}
	printf("\n");
}

int main(int argc, char *argv[]) {
	unsigned int N = 32; // size of vectors
	const unsigned int size = N * sizeof(int);

	int T = 32, B = 1; // threads per block and blocks per grid

	int *a, *b, *c; // host pointers
	int *dev_a, *dev_b, *dev_c; // device pointers to host memory


	// runtime must be placed into a state enabling to allocate zero-copy buffers.
	checkCuda(hipSetDeviceFlags(hipDeviceMapHost));

	checkCuda(
			hipHostAlloc((void**) &a, size,
					hipHostMallocWriteCombined | hipHostMallocMapped));
	checkCuda(
			hipHostAlloc((void**) &b, size,
					hipHostMallocWriteCombined | hipHostMallocMapped));

	checkCuda(hipHostAlloc((void**) &c, size, hipHostMallocMapped));

	// init vectors
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i;
	}
	printf("Vector A: \n");
	printVector(a, N);
	printf("Vector B: \n");
	printVector(b, N);

	// mem. copy to device not need now, but ptrs needed instead
	checkCuda(hipHostGetDevicePointer(&dev_a, a, 0));
	checkCuda(hipHostGetDevicePointer(&dev_b, b, 0));
	checkCuda(hipHostGetDevicePointer(&dev_c, c, 0));

	// to measure time
	hipEvent_t start, stop;
	float elapsed_time_ms;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	addVector<<<B, T>>>(dev_a, dev_b, dev_c, N);

	// copy back not needed but now need thread synchronization
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);

	// 	print results
	printf("Vector C: \n");
	printVector(c, N);

	hipEventElapsedTime(&elapsed_time_ms, start, stop);
	// print out execution time
	printf("Time to calculate results: %.2f ms.\n", elapsed_time_ms);

	// clean up
	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
