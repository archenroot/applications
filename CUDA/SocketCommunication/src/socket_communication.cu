#include "hip/hip_runtime.h"
/**
 * Licensed to the Apache Software Foundation (ASF) under one or more
 * contributor license agreements.  See the NOTICE file distributed with
 * this work for additional information regarding copyright ownership.
 * The ASF licenses this file to You under the Apache License, Version 2.0
 * (the "License"); you may not use this file except in compliance with
 * the License.  You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "util/cuPrintf.cu"

#include "hadoop/SerialUtils.hh"
#include "hadoop/StringUtils.hh"

#include <stdio.h>
#include <stdlib.h>

#include <signal.h>

#include <assert.h>
#include <errno.h>

#include <sys/types.h>
#include <netinet/in.h>
#include <sys/socket.h>

#include <iostream>
#include <fstream>

#include <stdint.h>

#include <string.h>
#include <strings.h>

#include <pthread.h>

#include <hip/hip_runtime.h>

/********************************************/
/*************** MESSAGE_TYPE ***************/
/********************************************/
enum MESSAGE_TYPE {
	GET_NEXT_VALUE, DONE
};

/********************************************/
/***************    Server    ***************/
/********************************************/
class SocketServer {
private:
	int sock;
	int port;
	bool done;

public:
	SocketServer() {
		sock = -1;
		port = -1;
		done = false;
	}

	~SocketServer() {
		fflush(stdout);
		if (sock != -1) {
			int result = shutdown(sock, SHUT_RDWR);
			//if (result != 0) {
			//	fprintf(stderr, "SocketServer: problem shutting socket\n");
			//}
			result = close(sock);
			if (result != 0) {
				fprintf(stderr, "SocketServer: problem closing socket\n");
			}
		}
	}

	int getPort() {
		return port;
	}

	void *runSocketServer() {
		printf("SocketServer started!\n");

		sock = socket(PF_INET, SOCK_STREAM, 0);
		if (sock < 0) {
			fprintf(stderr, "SocketServer: problem creating socket: %s\n",
					strerror(errno));
		}

		sockaddr_in addr;
		memset((char *) &addr, 0, sizeof(addr));
		addr.sin_family = AF_INET;
		// bind to a OS-assigned random port.
		addr.sin_port = htons(0);
		addr.sin_addr.s_addr = htonl(INADDR_LOOPBACK);

		int ret = bind(sock, (sockaddr*) &addr, sizeof(addr));
		if (ret < 0) {
			fprintf(stderr, "SocketServer: error on binding: %s\n",
					strerror(errno));
			return NULL;
		}

		// Get current port
		struct sockaddr_in current_addr;
		int current_addr_len = sizeof(current_addr);
		ret = getsockname(sock, (sockaddr*) &current_addr,
				(socklen_t *) &current_addr_len);
		if (ret < 0) {
			fprintf(stderr, "SocketServer: problem getsockname: %s\n",
					strerror(errno));
			return NULL;
		}
		port = ntohs(current_addr.sin_port);

		listen(sock, 1);

		printf("SocketServer is running @ port %d ...\n", port);

		sockaddr_in partnerAddr;
		int adrLen;
		int clientSock = accept(sock, (sockaddr*) &partnerAddr,
				(socklen_t *) &adrLen);

		printf("SocketServer: Client connected.\n");

		FILE* in_stream = fdopen(clientSock, "r");
		FILE* out_stream = fdopen(clientSock, "w");
		HadoopUtils::FileInStream* inStream = new HadoopUtils::FileInStream();
		HadoopUtils::FileOutStream* outStream =
				new HadoopUtils::FileOutStream();
		inStream->open(in_stream);
		outStream->open(out_stream);

		while (!done) {

			printf("SocketServer: wait for next command!\n");
			int32_t cmd = HadoopUtils::deserializeInt(*inStream);

			switch (cmd) {

			case GET_NEXT_VALUE: {
				int32_t val = HadoopUtils::deserializeInt(*inStream);
				HadoopUtils::serializeInt(val + 1, *outStream);
				outStream->flush();
				printf("SocketServer - GET_NEXT_VALUE IN=%d OUT=%d\n", val,
						val + 1);
				break;
			}
			case DONE: {
				printf("SocketServer - DONE\n");
				done = true;
				break;
			}

			default:
				fprintf(stderr, "SocketServer - Unknown binary command: %d\n",
						cmd);
				break;
			}
		}

		inStream->close();
		outStream->close();
		close(clientSock);

		delete inStream;
		delete outStream;

		printf("SocketServer stopped!\n");
		pthread_exit(0);
	}

	static void *SocketServer_thread(void *context) {
		return ((SocketServer *) context)->runSocketServer();
	}

}
;

/********************************************/
/**************     CLIENT     **************/
/********************************************/
class SocketClient {
private:
	int sock;
	FILE* in_stream;
	FILE* out_stream;
	HadoopUtils::FileInStream* inStream;
	HadoopUtils::FileOutStream* outStream;

public:
	SocketClient() {
		sock = -1;
		in_stream = NULL;
		out_stream = NULL;
	}

	void connectSocket(int port) {
		printf("SocketClient started\n");

		if (port <= 0) {
			printf("SocketClient: invalid port number!\n");
			return; /* Failed */
		}

		sock = socket(PF_INET, SOCK_STREAM, 0);
		if (sock == -1) {
			fprintf(stderr, "SocketClient: problem creating socket: %s\n",
					strerror(errno));
		}

		sockaddr_in addr;
		addr.sin_family = AF_INET;
		addr.sin_port = htons(port);
		addr.sin_addr.s_addr = htonl(INADDR_LOOPBACK);

		int res = connect(sock, (sockaddr*) &addr, sizeof(addr));
		if (res != 0) {
			fprintf(stderr,
					"SocketClient: problem connecting command socket: %s\n",
					strerror(errno));
		}

		in_stream = fdopen(sock, "r");
		out_stream = fdopen(sock, "w");

		inStream = new HadoopUtils::FileInStream();
		inStream->open(in_stream);
		outStream = new HadoopUtils::FileOutStream();
		outStream->open(out_stream);

		printf("SocketClient is connected to port %d ...\n", port);
	}

	~SocketClient() {
		if (in_stream != NULL) {
			fflush(in_stream);
		}
		if (out_stream != NULL) {
			fflush(out_stream);
		}
		fflush(stdout);
		if (sock != -1) {
			int result = shutdown(sock, SHUT_RDWR);
			if (result != 0) {
				fprintf(stderr, "SocketClient: problem shutting down socket\n");
			}
			result = close(sock);
			if (result != 0) {
				fprintf(stderr, "SocketClient: problem closing socket\n");
			}
		}
	}

	__device__ __host__ int getNextValue(int val) {

		HadoopUtils::serializeInt(GET_NEXT_VALUE, *outStream);
		HadoopUtils::serializeInt(val, *outStream);
		outStream->flush();

		int return_val = HadoopUtils::deserializeInt(*inStream);

		printf("SocketClient sent GET_NEXT_VALUE OUT=%d IN=%d\n", val,
				return_val);

		return return_val;
	}

	__device__ __host__ void sendDone() {

		HadoopUtils::serializeInt(DONE, *outStream);
		outStream->flush();
		printf("SocketClient sent DONE\n");
	}
};

// global vars
SocketServer socket_server;
pthread_t t_socket_server;
SocketClient *host_client;

void sigint_handler(int s) {
	printf("Caught signal %d\n", s);

	host_client->sendDone();

	pthread_join(t_socket_server, NULL);
	exit(0);
}

/********************************************/
/***************     CUDA     ***************/
/********************************************/

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}

__global__ void device_method(SocketClient *d_socket_client) {

	//d_socket_client

	//int val = d_object->getValue();
	//cuPrintf("Device object value: %d\n", val);
	//d_object->setValue(++val);
	//__threadfence();
}

int main(void) {

	// register SIGINT (STRG-C) handler
	struct sigaction sigIntHandler;
	sigIntHandler.sa_handler = sigint_handler;
	sigemptyset(&sigIntHandler.sa_mask);
	sigIntHandler.sa_flags = 0;
	sigaction(SIGINT, &sigIntHandler, NULL);

	// start socketServer
	pthread_create(&t_socket_server, NULL, &SocketServer::SocketServer_thread,
			&socket_server);

	// runtime must be placed into a state enabling to allocate zero-copy buffers.
	checkCuda(hipSetDeviceFlags(hipDeviceMapHost));

	// init host socket client as pinned memory
	checkCuda(
			hipHostAlloc((void**) &host_client, sizeof(SocketClient),
					hipHostMallocWriteCombined | hipHostMallocMapped));

	// connect SocketClient
	host_client->connectSocket(socket_server.getPort());

	int value = host_client->getNextValue(0);
	printf("Host client getNextValue: %d\n", value);

	SocketClient *device_client;
	checkCuda(hipHostGetDevicePointer(&device_client, host_client, 0));

	// initialize cuPrintf
	cudaPrintfInit();

	//device_method<<<1, 1>>>(device_client);
	//device_method<<<16, 4>>>(device_client);

	// display the device's output
	cudaPrintfDisplay();
	// clean up after cuPrintf
	cudaPrintfEnd();

	//printf("Host object value: %d (after gpu execution) (thread_num=%d)\n",
	//		host_client->getValue(), 16 * 4);

	//assert(host_client->getValue() == 16*4);

	sleep(2);

	host_client->sendDone();
	// wait for SocketServer
	pthread_join(t_socket_server, NULL);

	return 0;
}
