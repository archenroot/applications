#include "hip/hip_runtime.h"
/**
 * Licensed to the Apache Software Foundation (ASF) under one or more
 * contributor license agreements.  See the NOTICE file distributed with
 * this work for additional information regarding copyright ownership.
 * The ASF licenses this file to You under the Apache License, Version 2.0
 * (the "License"); you may not use this file except in compliance with
 * the License.  You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "util/cuPrintf.cu"

#include "socket/MessageType.hh"
#include "socket/SocketServer.hh"
#include "socket/SocketClient.hh"

#include <stdio.h>
#include <stdlib.h>

#include <signal.h>
#include <pthread.h>

#include <hip/hip_runtime.h>

// Global vars
pthread_t t_socket_server;
SocketServer socket_server;
SocketClient socket_client;

class KernelWrapper {
private:
	MESSAGE_TYPE command;
	int param1;
	bool result_available;
	int resultInt;
	//string resultString;
public:
	int active_thread_id;

	KernelWrapper() {
		active_thread_id = -1;
		result_available = false;
	}
	~KernelWrapper() {
	}

	__device__ __host__ int getValue(int val) {

		command = GET_NEXT_VALUE;
		param1 = val;

		while (!result_available) {
		}

		//d_object->setValue(++val);
		//__threadfence();

		return resultInt;
	}

	__device__ __host__ void done() {
		command = DONE;
	}
};

void sigint_handler(int s) {
	printf("Caught signal %d\n", s);

	socket_client.sendDone();

	pthread_join(t_socket_server, NULL);
	exit(0);
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}

__global__ void device_method(KernelWrapper *d_kernelWrapper) {

	int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	atomicExch(&d_kernelWrapper->active_thread_id, thread_id);

	if (d_kernelWrapper->active_thread_id == thread_id) {

		int val = d_kernelWrapper->getValue(thread_id);
		cuPrintf("Device object value: %d\n", val);
	}
}

int main(void) {

	// register SIGINT (STRG-C) handler
	struct sigaction sigIntHandler;
	sigIntHandler.sa_handler = sigint_handler;
	sigemptyset(&sigIntHandler.sa_mask);
	sigIntHandler.sa_flags = 0;
	sigaction(SIGINT, &sigIntHandler, NULL);

	// start SocketServer
	pthread_create(&t_socket_server, NULL, &SocketServer::thread,
			&socket_server);

	// wait for SocketServer to come up
	while (socket_server.getPort() == -1) {
	}

	// connect SocketClient
	socket_client.connectSocket(socket_server.getPort());

	int value = socket_client.getNextValue(0);
	printf("Host client getNextValue: %d\n", value);

	//CUDA setup
	// runtime must be placed into a state enabling to allocate zero-copy buffers.
	checkCuda(hipSetDeviceFlags(hipDeviceMapHost));

	KernelWrapper *h_kernelWrapper;
	KernelWrapper *d_kernelWrapper;

	// init host socket client as pinned memory
	checkCuda(
			hipHostAlloc((void**) &h_kernelWrapper, sizeof(KernelWrapper),
					hipHostMallocWriteCombined | hipHostMallocMapped));

	checkCuda(hipHostGetDevicePointer(&d_kernelWrapper, h_kernelWrapper, 0));

	// initialize cuPrintf
	cudaPrintfInit();

	//device_method<<<1, 1>>>(d_kernelWrapper);

	// display the device's output
	cudaPrintfDisplay();
	// clean up after cuPrintf
	cudaPrintfEnd();

	//printf("Host object value: %d (after gpu execution) (thread_num=%d)\n",
	//		host_client->getValue(), 16 * 4);

	//assert(host_client->getValue() == 16*4);

	sleep(2);

	socket_client.sendDone();
	// wait for SocketServer
	pthread_join(t_socket_server, NULL);

	return 0;
}
